#include "hip/hip_runtime.h"
/* 
@author: David L. George
@modified to C by: Brian Kyanjo
@date: 31 July 2023
@description: Solves normal Riemann problems for the 2D shallow water equations (swe) with 
topography:
            h_t + (hu)_x + (hv)_y = 0
            (hu)_t + (hu^2 + 1/2gh^2)_x + (huv)_y = -ghb_x
            (hv)_t + (huv)_x + (hv^2 + 1/2gh^2)_y = -ghb_y
where h is the height, u is the x velocity, v is the y velocity, g is the gravitational constant, and b is the topography.
@input: ql - conatins the state vector at the left edge of each cell
        qr - contains the state vector at the right edge of each cell
        
        This data is along a slice in the x-direction if idir = 0 or along a slice in the y-direction if idir = 1.

        idir - indicates the direction of the slice

@note: - The ith Riemann problem has left state qr(i-1,:) and right state ql(i,:).
       - This solver allows the user to easily select a Riemann solver in riemann_solvers.c,    this routine initializes all the variables for the swe, accounting for wet dry boundary, dry cells, wave speeds, etc.
       
@reference: David L. George
*/

#define maxiter 1

#include "../flood_speed_user.h"
#include "variables.h"
#include <math.h>
#include <fc2d_geoclaw.h>
#include <fc2d_cudaclaw_check.h>
#include <fc2d_cudaclaw_options.h>
#include <cudaclaw_user_fort.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>
#include <fclaw2d_include_all.h>

/* Extern declarations*/
extern __constant__ GeofloodVars d_geofloodVars;

/* function prototypes */
__device__ void riemanntype(double hL, double hR, double uL, double uR, double *hm, double *s1m, double *s2m, bool *rare1, bool *rare2);

__device__ void riemann_type(double hL, double hR, double uL, double uR, double hm, 
    double s1m, double s2m, bool rare1, bool rare2);

__device__ void riemann_aug_JCP(int meqn, int mwaves, double hL,
    double hR, double huL, double huR, double hvL, double hvR, 
    double bL, double bR, double uL, double uR, double vL, 
    double vR, double phiL, double phiR, double sE1, double sE2, double* sw, double* fw, int ix, int iy, int idir);

__device__ void riemann_aug_JCP(int meqn, int mwaves, double hL,
        double hR, double huL, double huR, double hvL, double hvR, 
        double bL, double bR, double uL, double uR, double vL, 
        double vR, double phiL, double phiR, double sE1, double sE2, double* sw1, double* sw2, double* sw3, double* fw11, double* fw12, double* fw13, double* fw21, double* fw22, double* fw23, double* fw31, double* fw32, double* fw33, int ix, int iy, int idir);

__device__ void flood_speed_compute_speeds(int idir, int meqn, int mwaves, int maux,
                                            double ql[], double  qr[],
                                            double auxl[], double auxr[],
                                            double s[])
{

    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double drytol = d_geofloodVars.dry_tolerance;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    int mcapa = d_geofloodVars.mcapa;

    int mu = 1+idir;
    // int mv = 2-idir;

    double hhat = (ql[0] + qr[0])/2.0;
    double hsq2 = sqrt(ql[0]) + sqrt(qr[0]);
    double uhat = (ql[mu]/sqrt(ql[0]) + qr[mu]/sqrt(qr[0]))/hsq2;
    // double vhat = (ql[mv]/sqrt(ql[0]) + qr[mv]/sqrt(qr[0]))/hsq2;
    double chat = sqrt(s_grav*hhat);

    // Roe wave speeds
    double roe1 = uhat - chat;
    double roe3 = uhat + chat;

    // left and right state wave speeds
    double s1l = ql[mu]/ql[0] - sqrt(s_grav*ql[0]);
    double s3r = qr[mu]/qr[0] + sqrt(s_grav*qr[0]);

    // Einfeldt wave speeds
    double s1 = fmin(s1l,roe1);
    double s3 = fmax(s3r,roe3);

    double s2 = 0.5*(s1+s3);

    s[0] = s1;
    s[1] = s2;
    s[2] = s3;
}

__device__ cudaclaw_cuda_speeds_t flood_speed_speeds = flood_speed_compute_speeds;

void flood_speed_assign_speeds(cudaclaw_cuda_speeds_t *speeds)
{
    hipError_t ce = hipMemcpyFromSymbol(speeds, HIP_SYMBOL(flood_speed_speeds), sizeof(cudaclaw_cuda_speeds_t));
    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (flood_speed_compute_speeds): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}

/* Normal Riemann solver for the 2d shallow water equations with topography */
// __device__ void cudaflood_rpn2(int idir, int meqn, int mwaves,
//                                 int maux, double ql[], double qr[],
//                                 double auxl[], double auxr[],
//                                 double fwave[], double s[], 
//                                 double amdq[], double apdq[], int ix, int iy)
__device__ void cudaflood_rpn2(int idir, int meqn, int mwaves,
                                int maux, double ql[], double qr[],
                                double auxl[], double auxr[],
                                double fwave[], double s[], 
                                double amd_q[], double apd_q[], int ix, int iy)
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double drytol = d_geofloodVars.dry_tolerance;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    int mcapa = d_geofloodVars.mcapa; 

    /* Local variables */
    double wall[3], fw[9], sw[3];
    double hR, hL, huR, huL, hvR, hvL, uR, uL, vR, vL, phiR, phiL;
    double bR, bL, sL, sR, sRoe1, sRoe2, sE1, sE2, uhat, chat;
    double hstar, hstartest, dxdc;
    double s1m, s2m;
    bool rare1, rare2;
    int mw, mu, mv;

    /* vectorized version */
    double fw11, fw12, fw13, fw21, fw22, fw23, fw31, fw32, fw33;
    double sw1, sw2, sw3;

    /* Swapping left to right  (cudaclaw_flux2.cu)*/
    // double *qr = q_l;
    // double *ql = q_r;
    // double *auxr = aux_l;
    // double *auxl = aux_r;
    double *amdq = apd_q;
    double *apdq = amd_q;

       //   print at only one thread
    //    int mx = 16, my = 16, mbc = 2;
    // //    int thread_index = threadIdx.x;
    //    int ifaces_x, ifaces_y;
    //   ifaces_x = mx + 2*mbc-1;
    //   ifaces_y = my + 2*mbc-1;
    //   int ix = thread_index % ifaces_x;
    //   int iy = thread_index/ifaces_x;  

      bool debug;
      if (idir == 0)
      {
        debug = 1;
      }
      else{
        debug = 0;
      }

    /* === Initializing === */
    /* inform of a bad riemann problem from the start */
    if ((qr[0] < 0.0) || (ql[0] < 0.0)) {
        printf("Negative input: hl, hr = %f,%f\n", ql[0], qr[0]);
    }

    // Initialize Riemann problem for the grid interface 
    for (mw=0; mw<mwaves; ++mw)
    {
        s[mw] = 0.0;
        fwave[mw + 0*mwaves] = 0.0;
        fwave[mw + 1*mwaves] = 0.0; 
        fwave[mw + 2*mwaves] = 0.0;
    }

    /* set normal direction */
    mu = 1+idir;
    mv = 2-idir;

    /* zero (small) negative values if they exist */
    // left state
    if (qr[0] < 0.0) {
        qr[0] = 0.0;
        qr[1] = 0.0;
        qr[2] = 0.0;
    }

    // right state
    if (ql[0] < 0.0) {
        ql[0] = 0.0;
        ql[1] = 0.0;
        ql[2] = 0.0;
    }

    // if (debug){
    //     printf("ix = %d, iy = %d\n " \ 
    //     "qr[0] = %.16f, ql[0] = %.16f\n" \
    //     "qr[1] = %.16f, ql[1] = %.16f\n" \
    //     "qr[2] = %.16f, ql[2] = %.16f\n\n", ix,iy,qr[0],ql[0],qr[1],ql[1],qr[2],ql[2]);
    // }

    // // Skip problem if in a completely dry area
    if (qr[0] <= drytol && ql[0] <= drytol) {
        goto label30;
    }

    // if (ql[0] > drytol || qr[0] > drytol) {
        /* Riemann problem variables */
        // hL  = qr[0];
        // hR  = ql[0];
        // huL = qr[mu];
        // huR = ql[mu];
        // bL = auxr[0];
        // bR = auxl[0];
        hL = ql[0];
        hR = qr[0];
        huL = ql[mu];
        huR = qr[mu];
        bL = auxl[0];
        bR = auxr[0];

        // hvL = qr[mv];
        // hvR = ql[mv];
        hvL = ql[mv];
        hvR = qr[mv];

        // Check for wet/dry left boundary
        if (hR > drytol) {
            uR = huR / hR;
            vR = hvR / hR;
            phiR = 0.5 * s_grav * (hR * hR) + (huR * huR) / hR;
        } else {
            hR = 0.0;
            huR = 0.0;
            hvR = 0.0;
            uR = 0.0;
            vR = 0.0;
            phiR = 0.0;
        }

        // Check for wet/dry right boundary
        if (hL > drytol) {
            uL = huL / hL;
            vL = hvL / hL;
            phiL = 0.5 * s_grav * (hL * hL) + (huL * huL) / hL;
        } else {
            hL = 0.0;
            huL = 0.0;
            hvL = 0.0;
            uL = 0.0;
            vL = 0.0;
            phiL = 0.0;
        }
    
        // if (debug){
        //     printf("ix = %d, iy = %d\n " \ 
        //     "hL = %.16f, hR = %.16f\n" \
        //     "huL = %.16f, huR = %.16f\n" \
        //     "hvL = %.16f, hvR = %.16f\n" \
        //     "uL = %.16f, uR = %.16f\n" \
        //     "vL = %.16f, vR = %.16f\n" \
        //     "phiL = %.16f, phiR = %.16f\n" \
        //     "bL = %.16f, bR = %.16f\n\n", ix,iy,hL,hR,huL,huR,hvL,hvR,uL,uR,vL,vR,phiL,phiR,bL,bR);
        // }

        /* left and right surfaces depth inrelation to topography */
        wall[0] = 1.0;
        wall[1] = 1.0;
        wall[2] = 1.0;
        if (hR <= drytol) {
            /* determine the wave structure */
            riemanntype(hL, hL, uL, -uL, &hstar, &s1m, &s2m, &rare1, &rare2);
            // riemann_type(hL, hL, -uL, uL, hstar, s1m, s2m, rare1, rare2);

        //     if (debug){
        //     printf("ix = %d, iy = %d\n " \ 
        //     "hL = %.16f, uL = %.16f\n" \
        //     "hstar = %.16f\n" \
        //     "s1m = %.16f, s2m = %.16f\n" \
        //     "rare1 = %d, rare2 = %d\n\n", ix,iy,hL,uL,hstar,s1m,s2m,rare1,rare2);
        // }

            hstartest = fmax(hL,hstar);
            if (hstartest + bL < bR) {
                /* hL+bL < bR and hstar+bL < bR, so water can't overtop right cell 
                (move into right cell) so right state should become ghost values 
                that mirror left for wall problem) */
                wall[1] = 0.0;
                wall[2] = 0.0;
                hR = hL;
                huR = -huL;
                bR = bL;
                phiR = phiL;
                uR = -uL;
                vR = vL;
                /* here we already have huR =- huL, so we don't need to change it */
            } else if (hL+bL < bR) {
                /* hL+bL < bR and hstar+bL >bR, so we set bR to the water level in 
                the left cell so that water can possibly overtop the right cell (move into the right cell) */ 
                bR = hL + bL;
            }
        } else if (hL <= drytol) { /* right surface is lower than left topo */
            /* determine the Riemann structure */
            riemanntype(hR, hR, -uR, uR, &hstar, &s1m, &s2m, &rare1, &rare2);
            // riemann_type(hR, hR, uR, -uR, hstar, s1m, s2m, rare1, rare2);
            hstartest = fmax(hR,hstar);

            // if (debug){
            //     printf("ix = %d, iy = %d\n " \ 
            //     "hR = %.16f, uR = %.16f\n" \
            //     "hstar = %.16f\n" \
            //     "s1m = %.16f, s2m = %.16f\n" \
            //     "rare1 = %d, rare2 = %d\n\n", ix,iy,hR,uR,hstar,s1m,s2m,rare1,rare2);
            // }

            if (hstartest + bR < bL) //left state should become ghost values that mirror right for wall problem
            {
                wall[0] = 0.0;
                wall[1] = 0.0;
                hL = hR;
                huL = -huR;
                bL = bR;
                phiL = phiR;
                uL = -uR;
                vL = vR;
            } else if (hR+bR < bL) {
                bL = hR + bR;
            }
        }

        // if (debug){
        //     printf("ix = %d, iy = %d\n " \ 
        //     "hL = %.16f, hR = %.16f\n" \
        //     "huL = %.16f, huR = %.16f\n" \
        //     "hvL = %.16f, hvR = %.16f\n" \
        //     "uL = %.16f, uR = %.16f\n" \
        //     "vL = %.16f, vR = %.16f\n" \
        //     "phiL = %.16f, phiR = %.16f\n" \
        //     "bL = %.16f, bR = %.16f\n\n", ix,iy,hL,hR,huL,huR,hvL,hvR,uL,uR,vL,vR,phiL,phiR,bL,bR);
        // }

        /* determine wave speeds */
        sL = uL - sqrt(s_grav*hL); // 1 wave speed of left state
        sR = uR + sqrt(s_grav*hR); // 2 wave speed of right state

        uhat = (sqrt(s_grav*hL)*uL + sqrt(s_grav*hR)*uR)/(sqrt(s_grav*hL) + sqrt(s_grav*hR)); // Roe average
        chat = sqrt(0.5*s_grav*(hL+hR)); // Roe average
        sRoe1 = uhat - chat; // Roe wave speed 1 wave
        sRoe2 = uhat + chat; // Roe wave speed 2 wave

        sE1 = fmin(sL,sRoe1); // Einfeldt wave speed 1 wave
        sE2 = fmax(sR,sRoe2); // Einfeldt wave speed 2 wave

        // if (ix == 7 && iy == 15) {
        //     if (debug){
        //         printf("ix = %d, iy = %d\n " \ 
        //         "sL = %.16f, sR = %.16f\n" \ 
        //         "sRoe1 = %.16f, sRoe2 = %.16f\n" \ 
        //         "sE1 = %.16f, sE2 = %.16f\n\n", ix,iy,sL,sR,sRoe1,sRoe2,sE1,sE2);
        //     }
        // }
        /* --- end of initializing --- */

        /* === solve Riemann problem === */
        // riemann_aug_JCP(meqn,mwaves,hL,hR,huL,huR,hvL,hvR,bL,bR,uL,uR,vL,vR,phiL,phiR,sE1,sE2,sw,fw,ix,iy,idir);
        riemann_aug_JCP(meqn,mwaves,hL,hR,huL,huR,hvL,hvR,bL,bR,uL,uR,vL,vR,phiL,phiR,sE1,sE2,&sw1,&sw2,&sw3,&fw11,&fw12,&fw13,&fw21,&fw22,&fw23,&fw31,&fw32,&fw33,ix,iy,idir);
        
        // Debugging check for NaNs 
        // if (tid == 0) {
        //     printf("hL = %e, hR = %e\n", hL, hR);
        //     printf("huL = %e, huR = %e\n", huL, huR);
        //     printf("hvL = %e, hvR = %e\n", hvL, hvR);
        //     printf("uL = %e, uR = %e\n", uL, uR);
        //     printf("vL = %e, vR = %e\n", vL, vR);
        //     printf("phiL = %e, phiR = %e\n", phiL, phiR);
        //     printf("bL = %e, bR = %e\n", bL, bR);
        // }


        // eliminate ghost fluxes for wall
        // for (mw=0; mw<3; mw++) {
        //     sw[mw] *= wall[mw];
        //     fw[mw] *= wall[mw];
        //     fw[mw + 1*mwaves] *= wall[mw];
        //     fw[mw + 2*mwaves] *= wall[mw];
        // }
        sw1 *= wall[0];
        sw2 *= wall[1];
        sw3 *= wall[2];
        fw11 *= wall[0];
        fw21 *= wall[0];
        fw31 *= wall[0];
        fw12 *= wall[1];
        fw22 *= wall[1];
        fw32 *= wall[1];
        fw13 *= wall[2];
        fw23 *= wall[2];
        fw33 *= wall[2];    
        
        // update fwave and corresponding speeds
        // for (mw=0; mw<mwaves; mw++) {
        //     s[mw] = sw[mw];
        //     fwave[mw] = fw[mw];
        //     fwave[mw + mu*mwaves] = fw[mw + 1*mwaves];
        //     fwave[mw + mv*mwaves] = fw[mw + 2*mwaves];
        // }
        s[0] = sw1;
        s[1] = sw2;
        s[2] = sw3;
        fwave[0] = fw11;
        fwave[1] = fw21;
        fwave[2] = fw31;
        fwave[3] = fw12;
        fwave[4] = fw22;
        fwave[5] = fw32;
        fwave[6] = fw13;
        fwave[7] = fw23;
        fwave[8] = fw33;

    // }

    // Debugging check for NaNs
    // if (tid == 0) {
    //     printf("s1 = %e, s2 = %e, s3 = %e\n", s[0], s[1], s[2]);
    //     printf("fwave[0] = %e, fwave[1] = %e, fwave[2] = %e\n", fwave[0], fwave[1], fwave[2]);
    //     printf("fwave[3] = %e, fwave[4] = %e, fwave[5] = %e\n", fwave[3], fwave[4], fwave[5]);
    //     printf("fwave[6] = %e, fwave[7] = %e, fwave[8] = %e\n", fwave[6], fwave[7], fwave[8]);
    // }

    label30: // (similar to 30 continue in Fortran)

    /* --- Capacity or Mapping from Latitude Longitude to physical space ----*/
    if (mcapa > 0) {
        if (idir == 0) {
            dxdc = earth_radius*deg2rad;
        } else {
            dxdc = earth_radius*cos(auxl[2])*deg2rad;
        }

        // update fwave and corresponding speeds
        for (mw=0; mw<mwaves; mw++) {
            s[mw] = dxdc*s[mw];
            fwave[mw] = dxdc*fwave[mw];
            fwave[mw + mwaves] = dxdc*fwave[mw + mwaves];
            fwave[mw + 2*mwaves] = dxdc*fwave[mw + 2*mwaves];
        }
    }

    /* --- compute fluctuations --- */
    for (mw=0; mw<mwaves; mw++) {
        if (s[mw] < 0.0) {
            amdq[mw] += fwave[mw];
            amdq[mw] += fwave[mw + 1*mwaves];
            amdq[mw] += fwave[mw + 2*mwaves];
        } else if (s[mw] > 0.0) {
            apdq[mw] += fwave[mw];
            apdq[mw] += fwave[mw + 1*mwaves];
            apdq[mw] += fwave[mw + 2*mwaves];
        } else {
            amdq[mw] += 0.5*fwave[mw];
            amdq[mw] += 0.5*fwave[mw + 1*mwaves];
            amdq[mw] += 0.5*fwave[mw + 2*mwaves];
            apdq[mw] += 0.5*fwave[mw];
            apdq[mw] += 0.5*fwave[mw + 1*mwaves];
            apdq[mw] += 0.5*fwave[mw + 2*mwaves];
        }
    }
    // Debugging check for NaNs
    // if (tid == 0) {
    //     printf("amdq[0] = %e, amdq[1] = %e, amdq[2] = %e\n", amdq[0], amdq[1], amdq[2]);
    //     printf("apdq[0] = %e, apdq[1] = %e, apdq[2] = %e\n", apdq[0], apdq[1], apdq[2]);
    // }
}


__device__ cudaclaw_cuda_rpn2_t flood_speed_rpn2 = cudaflood_rpn2;

void flood_speed_assign_rpn2(cudaclaw_cuda_rpn2_t *rpn2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpn2, HIP_SYMBOL(flood_speed_rpn2), sizeof(cudaclaw_cuda_rpn2_t));
    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (cudaflood_rpn2): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}

/* Transverse Riemann solver for the 2d shallow water equations with topography 
@desc: Using The Jacobian matrix from left cell (imp == 0) or right cell (imp == 1) to compute the transverse fluxes.
*/

__device__ void cudaflood_rpt2(int idir, int meqn, int mwaves, int maux,
                double q_l[], double q_r[], double aux1[], 
                double aux2[], double aux3[], int imp, 
                double asdq[], double bmasd_q[], double bpasd_q[], int ix, int iy) 
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double drytol = d_geofloodVars.dry_tolerance;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    int mcapa = d_geofloodVars.mcapa;

    int mw, mu, mv;
    double s[3], beta[3];
    double r[3][3];
    double h, u, v;
    double delf1, delf2, delf3;
    double dxdcm, dxdcp, topo1, topo3, eta;

    /* Swapping left to right  (cudaclaw_flux2.cu)*/
    double *qr = q_l;
    double *ql = q_r;
    double *bmasdq = bpasd_q;
    double *bpasdq = bmasd_q;

    mu = 1+idir;
    mv = 2-idir;

    h = (imp == 0) ? qr[0] : ql[0];

    bool debug = (idir == 0) ? 1 : 0;
  
    // if (h <= drytol) return; // skip problem if dry cell (leaves bmadsq(:) = bpasdq(:) = 0)
    if (h > drytol) {
        /* Compute velocities in relevant cell, and other quantities */
        if (imp == 0) {
            // fluctuations being split is left-going
            u = qr[mu] / h;
            v = qr[mv] / h;
            eta = h + aux2[0];
            topo1 = aux1[0];
            topo3 = aux3[0];
        } else {
            // fluctuations being split is right-going
            u = ql[mu] / h;
            v = ql[mv] / h;
            eta = h + aux2[0];
            topo1 = aux1[0];
            topo3 = aux3[0];
        }

        // Debugging check for NaNs
        // if (tid == 0) {
        //     printf("h = %e, u = %e, v = %e, eta = %e, topo1 = %e, topo3 = %e\n", h, u, v, eta, topo1, topo3);
        // }

        /* Check if cell that transverse wave go into are both too high: */
        // if (eta < fmin(topo1, topo3)) return; 
        if (eta >= fmin(topo1, topo3)) {

            /* Check if cell that transverse waves go into are both to high, if so,
            do the splitting (no dry cells), and compute necessary quantities */
            if (coordinate_system == 2) {
                // On the sphere
                if (idir == 1) {
                    dxdcp = earth_radius * deg2rad;
                    dxdcm = dxdcp;
                } else {
                    if (imp == 0) {
                        dxdcp = earth_radius * cos(aux3[2]) * deg2rad;
                        dxdcm = earth_radius * cos(aux1[2]) * deg2rad;
                    } else {
                        dxdcp = earth_radius * cos(aux3[2]) * deg2rad;
                        dxdcm = earth_radius * cos(aux1[2]) * deg2rad;
                    }
                }
            } else {
                // Cartesian
                dxdcp = 1.0;
                dxdcm = 1.0;
            }

            /* Compute some speeds necessary for the Jacobian 
            - Computing upgoing, downgoing waves either in cell on left (if imp==0)
                or on the right (if imp==1) 
            - To achieve this we use q values in cells above and below, however these
                aren't available (only in aux values)
            */
            s[0] = v - sqrt(s_grav * h);
            s[1] = v;
            s[2] = v + sqrt(s_grav * h);

            // Debugging check for NaNs
            if (ix == 7 && iy == 15) {
                // if ((hL >= 0.3280909317849093) && (hR >= 0.3280909317849093)){
                if (debug){
                    printf("ix = %d, iy = %d\n " \
                    "s[0] = %e, s[1] = %e, s[2] = %e\n", ix,iy, s[0], s[1], s[2]);
                }
            }

            /* Determine asdq decomposition (beta) */
            delf1 = asdq[0];
            delf2 = asdq[mu];
            delf3 = asdq[mv];

            // Debugging check for NaNs
            // if (debug) {
            //     printf("delf1 = %e, delf2 = %e, delf3 = %e\n", delf1, delf2, delf3);
            // }

            beta[0] = (s[2]*delf1 - delf3) / (s[2] - s[0]);
            beta[1] = -u*delf1 + delf2;
            beta[2] = (delf3 - s[0]*delf1) / (s[2] - s[0]);

            /* set-up eigenvectors */
            // r[0] = 1.0;
            // r[mu] = u;
            // r[mv] = s[0];
            r[0][0] = 1.0;
            r[1][0] = u;
            r[2][0] = s[0];

            // r[0 + mwaves] = 0.0;
            // r[mu + mwaves] = 1.0;
            // r[mv + mwaves] = 0.0;
            r[0][1] = 0.0;
            r[1][1] = 1.0;
            r[2][1] = 0.0;

            // r[0 + 2*mwaves] = 1.0;
            // r[mu + 2*mwaves] = u;
            // r[mv + 2*mwaves] = s[2];
            r[0][2] = 1.0;
            r[1][2] = u;
            r[2][2] = s[2];

            // Debugging check for NaNs
            // if (tid == 0) {

            //     printf("beta[0] = %e, beta[1] = %e, beta[2] = %e\n", beta[0], beta[1], beta[2]);
            // }

            /* Compute transverse fluctuations */
            for (mw = 0; mw < 3; mw++) {
                if ((s[mw] < 0.0) && (eta >= topo1)) {
                    // bmasdq[0] += dxdcm * s[mw]*beta[mw]*r[mw + mwaves];
                    // bmasdq[mu] += dxdcm * s[mw]*beta[mw]*r[mw + mwaves];
                    // bmasdq[mv] += dxdcm * s[mw]*beta[mw]*r[mw + 2*mwaves];
                    bmasdq[0] += dxdcm * s[mw]*beta[mw]*r[0][mw];
                    bmasdq[mu] += dxdcm * s[mw]*beta[mw]*r[1][mw];
                    bmasdq[mv] += dxdcm * s[mw]*beta[mw]*r[2][mw];
                } else if ((s[mw] > 0.0) && (eta >= topo3)) {
                    // bpasdq[0] += dxdcp * s[mw]*beta[mw]*r[mw + mwaves];
                    // bpasdq[mu] += dxdcp * s[mw]*beta[mw]*r[mw + mwaves];
                    // bpasdq[mv] += dxdcp * s[mw]*beta[mw]*r[mw + 2*mwaves];
                    bpasdq[0] += dxdcp * s[mw]*beta[mw]*r[0][mw];
                    bpasdq[mu] += dxdcp * s[mw]*beta[mw]*r[1][mw];
                    bpasdq[mv] += dxdcp * s[mw]*beta[mw]*r[2][mw];
                }
            }
            // Debugging check for NaNs
            // if (tid == 0) {
            //     printf("bmasdq[0] = %e, bmasdq[1] = %e, bmasdq[2] = %e\n", bmasdq[0], bmasdq[1], bmasdq[2]);
            //     printf("bpasdq[0] = %e, bpasdq[1] = %e, bpasdq[2] = %e\n", bpasdq[0], bpasdq[1], bpasdq[2]);
            // }
        }
    }
}



__device__ cudaclaw_cuda_rpt2_t flood_speed_rpt2 = cudaflood_rpt2;

void flood_speed_assign_rpt2(cudaclaw_cuda_rpt2_t *rpt2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpt2, HIP_SYMBOL(flood_speed_rpt2), sizeof(cudaclaw_cuda_rpt2_t));

    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (cudaflood_rpt2): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}

/* === Begin fuction riemann_aug_JCP======================================================== @description: - Solves swe give single left and right states
@note: - To use the original solver call with maxiter=1.
       - This solver allows iteration when maxiter > 1. The iteration seems to help  
         with instabilities that arise (with any solver) as flow becomes transcritical 
         over variable topography due to loss of hyperbolicity. 
*/

// __device__ void riemann_aug_JCP(int meqn, int mwaves, double hL,
//     double hR, double huL, double huR, double hvL, double hvR, 
//     double bL, double bR, double uL, double uR, double vL, 
//     double vR, double phiL, double phiR, double sE1, double sE2, double* sw, double* fw, int ix, int iy, int idir)
__device__ void riemann_aug_JCP(int meqn, int mwaves, double hL,
        double hR, double huL, double huR, double hvL, double hvR, 
        double bL, double bR, double uL, double uR, double vL, 
        double vR, double phiL, double phiR, double sE1, double sE2, double* sw1, double* sw2, double* sw3, double* fw11, double* fw12, double* fw13, double* fw21, double* fw22, double* fw23, double* fw31, double* fw32, double* fw33, int ix, int iy, int idir)
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double drytol = d_geofloodVars.dry_tolerance;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    int mcapa = d_geofloodVars.mcapa;

    /* Local variables */
    // double A[9], r[9], lambda[3], del[3], beta[3];
    double lambda[3], beta[3],del[3];
    double A[3][3], r[3][3];
    double delh, delhu, delphi, delb, delnorm;
    double rare1st, rare2st, sdelta, raremin, raremax;
    double criticaltol, convergencetol;
    double criticaltol_2, hustar_interface;
    double s1s2bar, s1s2tilde, hbar, hLstar, hRstar;
    double huRstar, huLstar, uRstar, uLstar, hstarHLL;
    double deldelh, deldelphi;
    double s1m, s2m, hm;
    double det1, det2, det3, determinant;
    bool rare1, rare2, rarecorrector, rarecorrectortest, sonic;
    int mw, k, iter;

    int mu = 1; // x-direction
    int mv = 2; // y-direction

    bool debug = (idir == 0) ? 1 : 0;

    /* determine del vectors */
    delh = hR - hL;
    delhu = huR - huL;
    delphi = phiR - phiL;
    delb = bR - bL;
    delnorm = delh * delh + delphi * delphi;

    /* Determine the Riemann structure */
    riemanntype(hL,hR,uL,uR,&hm,&s1m,&s2m,&rare1,&rare2);
    // riemann_type(hL,hR,uL,uR,hm,s1m,s2m,rare1,rare2);

    if (ix == 7 && iy == 15) {
        // if ((hL >= 0.3280909317849093) && (hR >= 0.3280909317849093)){
            if (debug){
                printf("ix = %d, iy = %d\n " \ 
                "hL = %.16f, hR = %.16f\n" \
                "uL = %.16f, uR = %.16f\n" \
                "hm = %.16f\n" \
                "s1m = %.16f, s2m = %.16f\n" \
                "g = %.16f, drytol = %.16f\n" \
                "rare1 = %d, rare2 = %d\n\n", ix,iy,hL,hR,uL,uR,hm,s1m,s2m,s_grav,drytol,rare1,rare2);
            }
        // }
    }
   
    /* For the solver to handle depth negativity, depth dh is included in the decompostion which gives as acess to using the depth positive semidefinite solver (HLLE). This makes the system to have 3 waves instead of 2. where the 1st and 3rd are the eigenpairs are related to the flux Jacobian matrix of the original SWE (since s1<s2<s3, and have been modified by Einfeldt to handle depth non-negativity) and the 2nd is refered to as the the entropy corrector wave since its introduced to correct entropy violating solutions with only 2 waves. */
    
    /* The 1st and 3rd speeds are the eigenvalues of the Jacobian matrix of the original SWE modified by Einfeldt's for use with the HLLE solver. */
    lambda[0] = fmin(sE1, s2m); /* sE1 - flux Jacobian eigen value s2m - Roe speed */
    lambda[2] = fmax(sE2, s1m); /* sE2 - flux Jacobian eigen value s1m - Roe speed*/

    /* Einfeldt's speeds */
    sE1 = lambda[0]; 
    sE2 = lambda[2];

    /* The 2nd speed is the entropy corrector wave speed. */
    lambda[1] = 0.0; /* no strong or significant rarefaction waves */
    
    /* determine the middle state in the HLLE solver */
    hstarHLL = fmax((hL*uL - hR*uR + (sE2 * hR) - (sE1 * hL)) / (sE2 - sE1), 0.0); /* middle state between the two discontinuities (positive semidefinite depth) */

    /* === determine the middle entropy corrector wave === */
    /* rarecorrectortest = .true. provides a more accurate Riemann solution but is more expensive. This is because a nonlinear Riemann solution with  2 nonlinear waves as a linear Riemann solution 3 (or 2 jump discontionuities to approximate 1 smooth nonlinear rarefaction if it's large). When rarecorrectortest = .false. the approximate solution has only 2 jump discontinuities instead of 3, so its less accurate but faster. */
    rarecorrectortest = false;
    rarecorrector = false;
    if (rarecorrectortest) {
        sdelta = lambda[2] - lambda[0];
        raremin = 0.5; /* indicate a large rarefaction wave but not large */
        raremax = 0.9; /* indicate a very large rarefaction wave */
       /* i.e (the total speed difference between the fastest and slowest wave in the Riemann solution = 0.5) */

        if (rare1 && sE1 * s1m < 0.0) raremin = 0.2;
        if (rare2 && sE2 * s2m < 0.0) raremin = 0.2;

        if (rare1 || rare2) {
            /* check which rarefaction is the strongest */
            rare1st = 3.0 * (sqrt(s_grav * hL) - sqrt(s_grav * hm));
            rare2st = 3.0 * (sqrt(s_grav * hR) - sqrt(s_grav * hm));
            if (fmax(rare1st, rare2st) > raremin * sdelta && fmax(rare1st, rare2st) < raremax * sdelta) {
                rarecorrector = true;
                if (rare1st > rare2st) {
                    lambda[1] = s1m;
                } else if (rare2st > rare1st) {
                    lambda[1] = s2m;
                } else {
                    lambda[1] = 0.5 * (s1m + s2m);
                }
            }
        }
        if (hstarHLL < fmin(hL, hR) / 5.0) rarecorrector = false;
    }

    /* determining modified eigen vectors */
    for (mw = 0; mw < mwaves; mw++) {   
        // r[mw] = 1.0; 
        // r[mw + mwaves] = lambda[mw]; 
        // r[mw + 2*mwaves] = pow(lambda[mw],2.0);
        r[0][mw] = 1.0;
        r[1][mw] = lambda[mw];
        r[2][mw] = pow(lambda[mw],2.0);
    }

    /* no strong rarefaction wave */
    if (!rarecorrector) {
        lambda[1]= 0.5*(lambda[0] + lambda[2]);
        // r[mwaves] = 0.0; // r[0,1]
        // r[mwaves + mu] = 0.0; // r[1,1]
        // r[mwaves + mv] = 1.0; // r[2,1]
        r[0][1] = 0.0;
        r[1][1] = 0.0;
        r[2][1] = 1.0;
    }

    /* === Determine the steady state wave === */
    criticaltol = fmax(drytol*s_grav, 1.0e-6);
    criticaltol_2 = sqrt(criticaltol);
    deldelh = -delb;
    deldelphi = -0.5 * (hR + hL) * (s_grav * delb); /* some approximation of the source term \int_{x_{l}}^{x_{r}} -g h b_x dx */

    /* determine a few quantities needed for steady state wave if iterated */
    hLstar = hL;
    hRstar = hR;
    uLstar = uL;
    uRstar = uR;
    huLstar = uLstar * hLstar;
    huRstar = uRstar * hRstar;

    /* iterate to better find the steady state wave */
    convergencetol = 1e-6;
    for (iter=1; iter <= maxiter; iter++) {
        /* determine steady state wave (this will be subtracted from the delta vectors */
        if (fmin(hLstar,hRstar) < drytol && rarecorrector) {
            rarecorrector = false;
            hLstar = hL;
            hRstar = hR;
            uLstar = uL;
            uRstar = uR;
            huLstar = uLstar*hLstar;
            huRstar = uRstar*hRstar;
            lambda[1] = 0.5*(lambda[0] + lambda[2]);
            // r[mwaves] = 0.0; // r[0,1]
            // r[mwaves + mu] = 0.0; // r[1,1]
            // r[mwaves + mv] = 1.0; // r[2,1]
            r[0][1] = 0.0;
            r[1][1] = 0.0;
            r[2][1] = 1.0;
        }

        /* For any two states; Q_i and Q_i-1, eigen values of SWE must satify: lambda(q_i)*lambda(q_i-1) = u^2 -gh, writing this conditon as a function of Q_i and Q_i-1, u and h become averages in lambda(q_i)*lambda(q_i-1) = u^2 -gh and these averages are denoted by bar and tilde. */
        hbar = fmax(0.5 * (hLstar + hRstar), 0.0);
        s1s2bar = 0.25 * pow((uLstar + uRstar),2) - (s_grav * hbar);
        s1s2tilde = fmax(0.0, uLstar * uRstar) - (s_grav * hbar);

        /* Based on the above conditon, smooth staedy state over slopping bathymetry cannot have a sonic point. Therefore, for regions with monotonically varying bathymetry, steady-state flow is either entirely subsonic (-u^2 +gh > 0) or entirely supersonic. */
        sonic = false;
        if (fabs(s1s2bar) <= criticaltol) {
            sonic = true;
        } else if (s1s2bar * s1s2tilde <= criticaltol * criticaltol) {
            sonic = true;
        } else if (s1s2bar * sE1 * sE2 <= criticaltol * criticaltol) {
            sonic = true;
        } else if (fmin(fabs(sE1), fabs(sE2)) < criticaltol_2) {
            sonic = true;
        } else if (sE1 < criticaltol_2 && s1m > -criticaltol_2) {
            sonic = true;
        } else if (sE2 > -criticaltol_2 && s2m < criticaltol_2) {
            sonic = true;
        } else if ((uL + sqrt(s_grav * hL)) * (uR + sqrt(s_grav * hR)) < 0.0) {
            sonic = true;
        } else if ((uL - sqrt(s_grav * hL)) * (uR - sqrt(s_grav * hR)) < 0.0) {
            sonic = true;
        }

        /* find jump in h, deldelh */
        if (sonic) {
            deldelh = -delb;
        } else {
            deldelh = delb * s_grav * hbar / s1s2bar;
        }

        /* find bounds in case of critical state resonance, or negative states */
        if (sE1 < -criticaltol && sE2 > criticaltol) {
            deldelh = fmin(deldelh, hstarHLL * (sE2 - sE1) / sE2);
            deldelh = fmax(deldelh, hstarHLL * (sE2 - sE1) / sE1);
        } else if (sE1 >= criticaltol) {
            deldelh = fmin(deldelh, hstarHLL * (sE2 - sE1) / sE1);
            deldelh = fmax(deldelh, -hL);
        } else if (sE2 <= -criticaltol) {
            deldelh = fmin(deldelh, hR);
            deldelh = fmax(deldelh, hstarHLL * (sE2 - sE1) / sE2);
        }

        /* find jump in phi, ddphi */
        if (sonic) {
            deldelphi = -s_grav * hbar * delb;
        } else {
            deldelphi = -delb * s_grav * hbar * s1s2tilde / s1s2bar;
        }

        /* find bounds in case of critical state resonance, or negative states */
        deldelphi = fmin(deldelphi, s_grav * fmax(-hLstar * delb, -hRstar * delb));
        deldelphi = fmax(deldelphi, s_grav * fmin(-hLstar * delb, -hRstar * delb));

        /* determine the delta vectors */
        del[0] = delh - deldelh;
        del[1] = delhu;
        del[2] = delphi - deldelphi;  

        /* Determine coefficients beta(k) using crammer's rule
          first determine the determinant of the eigenvector matrix */
        // det1 = r[0]*(r[mwaves + mu]*r[2*mwaves + mv] - r[2*mwaves + mu]*r[mwaves + mv]);
        // det2 = r[mwaves]*(r[mu]*r[2*mwaves + mv] - r[2*mwaves + mu]*r[mv]);
        // det3 = r[2*mwaves]*(r[mu]*r[mwaves + mv] - r[mwaves + mu]*r[mv]);
        det1 = r[0][0]*(r[1][1]*r[2][2] - r[1][2]*r[2][1]);
        det2 = r[0][1]*(r[1][0]*r[2][2] - r[1][2]*r[2][0]);
        det3 = r[0][2]*(r[1][0]*r[2][1] - r[1][1]*r[2][0]);
        determinant = det1 - det2 + det3;

        /* solve for beta(k) */
        for(k=0; k < 3; k++)
        {   
            for(mw=0; mw < 3; mw++)
            {
                // A[mw] = r[mw]; 
                // A[mw + mwaves] = r[mw + mwaves];
                // A[mw + 2*mwaves] = r[mw + 2*mwaves];
                A[0][mw] = r[0][mw];
                A[1][mw] = r[1][mw];
                A[2][mw] = r[2][mw];
            }
            // A[k] = del[0];
            // A[mwaves + k] = del[1];
            // A[2*mwaves + k] = del[2];
            // det1 = A[0]*(A[mwaves + mu]*A[2*mwaves + mv] - A[2*mwaves + mu]*A[mwaves + mv]);
            // det2 = A[mwaves]*(A[mu]*A[2*mwaves + mv] - A[2*mwaves + mu]*A[mv]);
            // det3 = A[2*mwaves]*(A[mu]*A[mwaves + mv] - A[mwaves + mu]*A[mv]);
            A[0][k] = del[0];
            A[1][k] = del[1];
            A[2][k] = del[2];
            det1 = A[0][0]*(A[1][1]*A[2][2] - A[1][2]*A[2][1]);
            det2 = A[0][1]*(A[1][0]*A[2][2] - A[1][2]*A[2][0]);
            det3 = A[0][2]*(A[1][0]*A[2][1] - A[1][1]*A[2][0]);
            beta[k] = (det1 - det2 + det3)/determinant;
        }

        /* exit if things aren't changing */
        if (fabs(pow(del[0],2)+pow(del[2],2.0) - delnorm) < convergencetol) break;

        delnorm = pow(del[0],2)+pow(del[2],2.0); /* update delnorm */

        /* find new states qLstar and qRstar on either side of interface */
        hLstar = hL;
        hRstar = hR;
        uLstar = uL;
        uRstar = uR;
        huLstar = uLstar*hLstar;
        huRstar = uRstar*hRstar;

        /* left state depth and momentum updates */
        for (mw=0; mw < mwaves; mw++)
        {
            if (lambda[mw] < 0.0)
            {
                // hLstar = hLstar + beta[mw]*r[mw]; 
                // huLstar = huLstar + beta[mw]*r[mw + mwaves]; 
               hLstar = hLstar + beta[mw]*r[0][mw];
               huLstar = huLstar + beta[mw]*r[1][mw];
            }
        }

        /* right state depth and momentum updates */
        for (mw = mwaves-1; mw >= 0; mw--)
        {
            if (lambda[mw] > 0.0)
            {
                // hRstar = hRstar - beta[mw]*r[mw]; 
                // huRstar = huRstar - beta[mw]*r[mw + mwaves]; 
                hRstar = hRstar - beta[mw]*r[0][mw];
                huRstar = huRstar - beta[mw]*r[1][mw];
            }
        }

        /* left state velocity update */
        if (hLstar > drytol) 
        {
            uLstar = huLstar/hLstar;
        }
        else  /* dry state */
        {
            hLstar = fmax(hLstar,0.0);
            uLstar = 0.0;
        }

        /* right state velocity update */
        if (hRstar > drytol) 
        {
            uRstar = huRstar/hRstar;
        }
        else /* dry state */
        {
            hRstar = fmax(hRstar,0.0);
            uRstar = 0.0;
        }
    } /* end of  iteration on the Riemann problem*/

    /* === determine the fwaves and speeds=== */
    // for (mw=0; mw < mwaves; mw++)
    // {
    //     sw[mw] = lambda[mw];
    //     // fw[mw] = beta[mw]*r[mw + mwaves]; 
    //     // fw[mw + mwaves] = beta[mw]*r[mw + 2*mwaves]; 
    //     // fw[mw + 2*mwaves] = beta[mw]*r[mw + mwaves]; 
    //     fw[mw] = beta[mw]*r[1][mw];
    //     fw[mw + mwaves] = beta[mw]*r[2][mw];
    //     fw[mw + 2*mwaves] = beta[mw]*r[1][mw];
    // }
    *sw1 = lambda[0];
    *sw2 = lambda[1];
    *sw3 = lambda[2];
    // mw = 0;
    *fw11 = beta[0]*r[1][0];
    *fw21 = beta[0]*r[2][0];
    *fw31 = beta[0]*r[1][0];
    // mw = 1;
    *fw12 = beta[1]*r[1][1];
    *fw22 = beta[1]*r[2][1];
    *fw32 = beta[1]*r[1][1];
    // mw = 2;
    *fw13 = beta[2]*r[1][2];
    *fw23 = beta[2]*r[2][2];
    *fw33 = beta[2]*r[1][2];

    // find transverse components (ie huv jumps)
    // fw[mv] *= vL;
    // fw[2*mwaves + mv] *= vR;
    // fw[mwaves + mv] = 0.0;
    *fw31 *= vL;
    *fw33 *= vR;
    *fw32 = 0.0;

    // hustar_interface = hL*uL + fw[0];
    // if (hustar_interface <= 0.0) {
    //     fw[mv] += (hR * uR * vR - hL * uL * vL - fw[mv] - fw[2*mwaves + mv]);
    // } else {
    //     fw[2*mwaves + mv] += (hR * uR * vR - hL * uL * vL - fw[mv] - fw[2*mwaves + mv]);
    // }
    hustar_interface = hL*uL + *fw11;
    if (hustar_interface <= 0.0) {
        *fw31 += (hR * uR * vR - hL * uL * vL - *fw31 - *fw33);
    } else {
        *fw33 += (hR * uR * vR - hL * uL * vL - *fw31 - *fw33);
    }

}

/* === Begin fuction Riemann type ============
 @description: Determines the Riemann structure (wave-type in each family)
*/

__device__ void riemanntype(double hL, double hR, double uL, double uR, double *hm, 
                            double *s1m, double *s2m, bool *rare1, bool *rare2)
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double drytol = d_geofloodVars.dry_tolerance;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    int mcapa = d_geofloodVars.mcapa;

    // Local variables
    double um, u1m, u2m, h0, F_max, F_min, dfdh, F0, slope, gL, gR;
    double sqrtgh1, sqrtgh2;
    int iter; 

    // Test for Riemann structure
    double h_min = fmin(hR,hL);
    double h_max = fmax(hR,hL);
    double delu = uR - uL;

    /* Have dry state on either side 
    - Only one rarefaction wave
    - another shock wave has 0 jump and moves at the same speed as one edge of the    rarefaction  wave */
    if (h_min <= drytol)
    {
        *hm = 0.0;
        um = 0.0;
    
        /* Either hR or hL is almost zero, so the expression below corresponds
           to either Eqn. (54a) or Eqn. (54b) in the JCP paper */
        *s1m = uR + uL - 2.0 * sqrt(s_grav * hR) + 2.0 * sqrt(s_grav * hL);
        *s2m = *s1m; 
        *rare1 = (hL <= 0.0) ? false : true;
        *rare2 = !(*rare1);
    } else {
        F_min = delu + 2.0 * (sqrt(s_grav * h_min) - sqrt(s_grav * h_max));
        F_max = delu + (h_max - h_min) * sqrt(0.5 * s_grav * (h_max + h_min) / (h_max * h_min));

        if (F_min > 0.0){  // 2-rarefactions
            /* Eqn (13.56) in the FVMHP book */
            *hm = (1.0 / (16.0 * s_grav)) * pow(fmax(0.0, -delu + 2.0 * (sqrt(s_grav * hL) + sqrt(s_grav * hR))), 2);
            um = copysign(1.0, *hm) * (uL + 2.0 * (sqrt(s_grav * hL) - sqrt(s_grav * *hm)));
            *s1m = uL + 2.0 * sqrt(s_grav * hL) - 3.0 * sqrt(s_grav * *hm);
            *s2m = uR - 2.0 * sqrt(s_grav * hR) + 3.0 * sqrt(s_grav * *hm);
            *rare1 = true;
            *rare2 = true;
        } else if (F_max <= 0.0) { // 2-shocks
            /* Below it solves for the intersection of two Hugoniot loci to get the
            accurate Riemann solution */
            /* Root finding using a Newton iteration on sqrt(h) */
            h0 = h_max;
            for (iter = 1; iter <= maxiter; iter++) {
                gL = sqrt(0.5 * s_grav * (1.0 / h0 + 1.0 / hL));
                gR = sqrt(0.5 * s_grav * (1.0 / h0 + 1.0 / hR));
                F0 = delu + (h0 - hL) * gL + (h0 - hR) * gR;
                dfdh = gL - s_grav * (h0 - hL) / (4.0 * h0 * h0 * gL) + gR - s_grav * (h0 - hR) / (4.0 * h0 * h0 * gR);
                slope = 2.0 * sqrt(h0) * dfdh;
                h0 = pow(sqrt(h0) - F0 / slope, 2);
            }
            *hm = h0;
            /* u1m and u2m are Eqns (13.19) and (13.20) in the FVMHP book */
            u1m = uL - (*hm - hL) * sqrt(0.5 * s_grav * (1.0 / *hm + 1.0 / hL));
            u2m = uR + (*hm - hR) * sqrt(0.5 * s_grav * (1.0 / *hm + 1.0 / hR));
            um = 0.5 * (u1m + u2m);
            *s1m = u1m - sqrt(s_grav * *hm);
            *s2m = u2m + sqrt(s_grav * *hm);
            *rare1 = false;
            *rare2 = false;
        } else { // 1-shock or 1-rarefaction
            h0 = h_min;
            for (iter = 1; iter <= maxiter; iter++) {
                F0 = delu + 2.0 * (sqrt(s_grav * h0) - sqrt(s_grav * h_max)) + (h0 - h_min) * sqrt(0.5 * s_grav * (1.0 / h0 + 1.0 / h_min));
                slope = (F_max - F0) / (h_max - h_min);
                h0 = h0 - F0 / slope;
            }
            *hm = h0;
            sqrtgh2 = sqrt(s_grav * *hm);
            if (hL > hR) {
                sqrtgh1 = sqrt(s_grav * hL);
                /* Eqn (13.55) in the FVMHP book */
                um = uL + 2.0 * sqrtgh1 - 2.0 * sqrtgh2;
                *s1m = uL + 2.0 * sqrtgh1 - 3.0 * sqrtgh2;
                *s2m = uL + 2.0 * sqrtgh1 - sqrtgh2;

                *rare1 = true;
                *rare2 = false;
            } else {
                sqrtgh1 = sqrt(s_grav * hR);
                um = uR - 2.0 * sqrtgh1 + 2.0 * sqrtgh2;
                *s1m = uR - 2.0 * sqrtgh1 + sqrtgh2;
                *s2m = uR - 2.0 * sqrtgh1 + 3.0 * sqrtgh2;
                *rare1 = false;
                *rare2 = true;
            }
        }
    }
} /* End of riemanntype function */



__device__  void riemann_type(double hL, double hR, double uL, double uR, double hm, 
    double s1m, double s2m, bool rare1, bool rare2)
{
    double g = d_geofloodVars.gravity;
    double drytol = d_geofloodVars.dry_tolerance;

    double um,u1m,u2m,delu;
    double h_max,h_min,h0,F_max,F_min,dfdh,F0,slope,gL,gR;
    double sqrtgh1,sqrtgh2;
    int iter;

    /* Test for Riemann structure */
    h_min = min(hR,hL);
    h_max = max(hR,hL);
    delu = uR - uL;

   if (h_min <= drytol){
    hm = 0.0;
    um = 0.0;
    s1m = uR + uL - 2.0*sqrt(g*hR) + 2.0*sqrt(g*hL);
    s2m = uR + uL - 2.0*sqrt(g*hR) + 2.0*sqrt(g*hL);

    if (hL <= 0.0){
        rare1 = false;
        rare2 = true;
    } else {
        rare1 = true;
        rare2 = false;
    }
   } else {
        F_min = delu + 2.0*(sqrt(g*h_min) - sqrt(g*h_max));
        F_max = delu + (h_max - h_min)*(sqrt(0.5*g*(h_max + h_min)/(h_max*h_min)));

        if (F_min < 0.0) {
            /* 2-rarefactions */
            hm = (1.0/(16.0*g))*(pow(max(0.0,-delu + 2.0*(sqrt(g*hL) + sqrt(g*hR))),2));
            um = copysign(1.0,hm)*(uL + 2.0*(sqrt(g*hL) - sqrt(g*hm)));
            s1m = uL + 2.0*sqrt(g*hL) - 3.0*sqrt(g*hm);
            s2m = uR - 2.0*sqrt(g*hR) + 3.0*sqrt(g*hm);
            rare1 = true;
            rare2 = true;
        } else if (F_max <= 0.0) {
            /* 2-shocks */
            /* Root finding using a Newton iteration on sqrt(h) */
            h0 = h_max;
            for (iter = 1; iter <= maxiter; iter++) {
                gL = sqrt(0.5*g*(1.0/h0 + 1.0/hL));
                gR = sqrt(0.5*g*(1.0/h0 + 1.0/hR));
                F0 = delu + (h0 - hL)*gL + (h0 - hR)*gR;
                dfdh = gL - g*(h0 - hL)/(4.0*h0*h0*gL) + gR - g*(h0 - hR)/(4.0*h0*h0*gR);
                slope = 2.0*sqrt(h0)*dfdh;
                h0 = pow(sqrt(h0) - F0/slope,2);
            }
            hm = h0;
            /* u1m and u2m are Eqns (13.19) and (13.20) in the FVMHP book */
            u1m = uL - (hm - hL)*sqrt(0.5*g*(1.0/hm + 1.0/hL));
            u2m = uR + (hm - hR)*sqrt(0.5*g*(1.0/hm + 1.0/hR));
            um = 0.5*(u1m + u2m);
            s1m = u1m - sqrt(g*hm);
            s2m = u2m + sqrt(g*hm);
            rare1 = false;
            rare2 = false;
        } else {
            /* 1-shock or 1-rarefaction */
            h0 = h_min;
            for (iter = 1; iter <= maxiter; iter++) {
                F0 = delu + 2.0*(sqrt(g*h0) - sqrt(g*h_max)) + (h0 - h_min)*sqrt(0.5*g*(1.0/h0 + 1.0/h_min));
                slope = (F_max - F0)/(h_max - h_min);
                h0 = h0 - F0/slope;
            }
            hm = h0;
            sqrtgh2 = sqrt(g*hm);
            if (hL > hR) {
                sqrtgh1 = sqrt(g*hL);
                /* Eqn (13.55) in the FVMHP book */
                um = uL + 2.0*sqrtgh1 - 2.0*sqrtgh2;
                s1m = uL + 2.0*sqrtgh1 - 3.0*sqrtgh2;
                s2m = uL + 2.0*sqrtgh1 - sqrtgh2;

                rare1 = true;
                rare2 = false;
            } else {
                sqrtgh1 = sqrt(g*hR);
                um = uR - 2.0*sqrtgh1 + 2.0*sqrtgh2;
                s1m = uR - 2.0*sqrtgh1 + sqrtgh2;
                s2m = uR - 2.0*sqrtgh1 + 3.0*sqrtgh2;
                rare1 = false;
                rare2 = true;
            }
        }
   }

}