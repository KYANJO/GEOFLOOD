#include "hip/hip_runtime.h"
/* 
@author: Brian Kyanjo
@date: 31 July 2023
@description: Solves transvere Riemann problems for the 2D shallow water equations (swe) with 
topography
*/

#include "geoflood_riemann_utils.h"
 
// --remember to call these variables from geoclaw_module.f90 ---//
__constant__ double grav;
__constant__ double dry_tolerance;
__constant__ double earth_radius;
__constant__ double deg2rad;
__constant__ int coordinate_system;

__device__ double fmax(double, double)
__device__ double fmin(double x, double y)
__device__ double fabs(double x)
__device__ double pow(double x, double y)
__device__ double sqrt(double x)
__device__ double sin(double x)
__device__ double cos(double x)


__device__ void cudaflood_rpt2(int idir, int meqn, int mwaves, int maux,
                                double ql[], double qr[], double aux1[], 
                                double aux2[], double aux3[], int imp, 
                                double asdq[], double bmasdq[], double bpasdq[]) //<-- added imp, don't forget to update it in cudaclaw_flux2.cu
{
    int i,m,mw,mu,mv;

    double s[2];
    double r[2][2];
    double beta[2];
    double abs_tol;
    double hl,hr,hul,hur,hvl,hvr,vl,vr,ul,ur,bl,br;
    double uhat,vhat,hhat,roe1,roe3,s1,s2,s3,s1l,s3r;
    double delf1,delf2,delf3,dxdcd,dxdcu;
    double dxdcm,dxdcp,topo1,topo3,eta;

    abs_tol = dry_tolerance;

    mu = 1+idir;
    mv = 2-idir;

    hl = qr[0];
    hr = ql[0];
    hul = qr[mu];
    hur = ql[mu];
    hvl = qr[mv];
    hvr = ql[mv];

    //--- determine velocity from momentum ---//
    if (hl < abs_tol)
    {   
        hl = 0.0;
        ul = 0.0;
        vl = 0.0;
    }
    else
    {
        ul = hul/hl;
        vl = hvl/hl;
    }

    if (hr < abs_tol)
    {
        hr = 0.0;
        ur = 0.0;
        vr = 0.0;
    }
    else
    {
        ur = hur/hr;
        vr = hvr/hr;
    }
    
    for (mw=0; mw < mwaves; mw++)
    {
        s[mw] = 0.0;
        beta[mw] = 0.0;
        for (m=0; m<meqn; m++)
        {
            r[mw][m] = 0.0;
        }
    }
    dxdcp = 1.0;
    dxdcm = 1.0;

    if (hl <= abs_tol && hr <= abs_tol) continue;

    // check and see if cell that transverse waves are going in is high and dry
    // if (imp == 0)
    // {
        eta = qr[0] + aux2[0];
        topo1 = aux1[0];
        topo3 = aux3[0];
    // }
    // else
    // {
    //     eta = ql[0] + aux2[0];
    //     topo1 = aux1[0];
    //     topo3 = aux3[0];
    // }
    if (eta < fmax(topo1,topo3)) continue;

    if (coordinate_system == 1)
    {
        if (ixy == 1 )
        {
            dxdcp = (earth_radius*deg2rad);
            dxdcm = dxdcp;
        }
        else
        {
            // if (imp == 0)
            // {
                dxdcp = earth_radius*cos(aux3[2])*deg2rad;
                dxdcm = earth_radius*cos(aux1[2])*deg2rad;
            // }
            // else
            // {
            //     dxdcp = earth_radius*cos(aux3[2])*deg2rad;
            //     dxdcm = earth_radius*cos(aux1[2])*deg2rad;
            // }
        }
    }
    // ---- determine some speeds necessary for the Jacobian ----//
    vhat = (vr*sqrt(hr))/(sqrt(hr)+sqrt(hl)) + (vl*sqrt(hl))/(sqrt(hr)+sqrt(hl));
    uhat = (ur*sqrt(hr))/(sqrt(hr)+sqrt(hl)) + (ul*sqrt(hl))/(sqrt(hr)+sqrt(hl));
    hhat = (hr + hl)/2.0;

    roe1 = vhat - sqrt(grav*hhat);
    roe3 = vhat + sqrt(grav*hhat);

    s1l = vl - sqrt(grav*hl);
    s3r = vr + sqrt(grav*hr);

    s1 = fmin(roe1,s1l);
    s3 = fmax(roe3,s3r);

    s2 = 0.5*(s1+s3);

    s[0] = s1;
    s[1] = s2;
    s[2] = s3;

    // ---- determine asdq decomposition (beta) ----//
    delf1 = asdq[0];
    delf2 = asdq[mu];
    delf3 = asdq[mv];

    beta[0] = (s3*delf1/(s3-s1)) - (delf3/(s3-s1));
    beta[1] = -s2*delf1 + delf2;
    beta[2] = (delf3/(s3-s1)) - (s1*delf1/(s3-s1));

    // --- Set-up eigenvectors matrix (r) ---//
    r[0][0] = 1.0;
    r[1][0] = s2;
    r[2][0] = s1;

    r[0][1] = 0.0;
    r[1][1] = 1.0;
    r[2][1] = 0.0;

    r[0][2] = 1.0;
    r[1][2] = s2;
    r[2][2] = s3;

    // ---- Compute fluctuations ----//
    bmasdq[0] = 0.0;
    bpasdq[0] = 0.0;
    bmasdq[1] = 0.0;
    bpasdq[1] = 0.0;
    bmasdq[2] = 0.0;
    bpasdq[2] = 0.0;
    for (mw=0; mw < mwaves; mw++)
    {
        if (sw[mw] < 0.0)
        {
            bmasdq[0] = bmasdq[0] + dxdcm*s[mw]*beta[mw]*r[0][mw];
            bmasdq[mu] = bmasdq[mu] + dxdcm*s[mw]*beta[mw]*r[1][mw];
            bmasdq[mv] = bmasdq[mv] + dxdcm*s[mw]*beta[mw]*r[2][mw];
        }
        else if (s[mw] > 0.0)
        {
            bpasdq[0] = bpasdq[0] + dxdcp*s[mw]*beta[mw]*r[0][mw];
            bpasdq[mu] = bpasdq[mu] + dxdcp*s[mw]*beta[mw]*r[1][mw];
            bpasdq[mv] = bpasdq[mv] + dxdcp*s[mw]*beta[mw]*r[2][mw];
        }
    }
}

__device__ cudaclaw_cuda_rpt2_t geoflood_rpt2 = cudaflood_rpt2;

void geoflood_assign_rpt2(cudaclaw_cuda_rpt2_t *rpt2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpt2, HIP_SYMBOL(geoflood_rpt2), sizeof(cudaclaw_cuda_rpt2_t));

    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (cudaflood_rpt2): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}