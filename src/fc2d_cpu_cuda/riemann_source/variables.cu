#include "../fc2d_cudaclaw_cuda.h"
#include "../fc2d_geoclaw_fort.h"
#include "variables.h"
#include <math.h>
#include <fc2d_cudaclaw_check.h>

/* Declare constant memory variables */
__constant__ GeofloodVars d_geofloodVars;

void setprob_cuda() {

    /*=== declare variables === */
    int coord_system_, num_manning_, friction_index_;
    // double dry_tol_;
    // int mcapa_;
    double grav_, earth_rad_, deg2rad_;
    double theta_0_, omega_, friction_depth_;
    bool coriolis_forcing_, friction_forcing_, variable_friction_;
    // double *manning_coeff_, *manning_break_;
    double manning_coeff_, manning_break_;

    GET_GEOCLAW_PARAMETERS(&coord_system_,&grav_, &earth_rad_,&deg2rad_, 
                           &theta_0_, &omega_, &coriolis_forcing_, &friction_forcing_, &friction_depth_,
                           &variable_friction_, &num_manning_, &friction_index_, &manning_coeff_, &manning_break_);

     /* === Create and populate structures on the host === */
    GeofloodVars geofloodVars;
    geofloodVars.gravity = grav_;
    // geofloodVars.dry_tolerance = dry_tol_;
    geofloodVars.earth_radius = earth_rad_;
    geofloodVars.coordinate_system = coord_system_;
    // geofloodVars.mcapa = mcapa_;
    geofloodVars.deg2rad = deg2rad_;
    geofloodVars.theta_0 = theta_0_;
    geofloodVars.omega = omega_;
    geofloodVars.coriolis_forcing = coriolis_forcing_;
    geofloodVars.friction_forcing = friction_forcing_;
    geofloodVars.friction_depth = friction_depth_;
    geofloodVars.variable_friction = variable_friction_;
    geofloodVars.num_manning = num_manning_;
    geofloodVars.friction_index = friction_index_;
    geofloodVars.manning_coefficent = manning_coeff_;
    geofloodVars.manning_break = manning_break_;

    /* === Copy structures to device (constant memory) === */
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_geofloodVars), &geofloodVars, sizeof(GeofloodVars)));

}
