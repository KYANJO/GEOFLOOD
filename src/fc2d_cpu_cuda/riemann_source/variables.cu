#include "../fc2d_cudaclaw_cuda.h"
#include "../fc2d_geoclaw_fort.h"
#include "variables.h"
#include <math.h>
#include <fc2d_cudaclaw_check.h>

/* Declare constant memory variables */
// __constant__ GeofloodVars d_geofloodVars;

// void setprob_cuda(){
//     int i = 0;
//     char * line = NULL, *p = NULL, *eptr;
//     size_t len = 0;
//     ssize_t read;
//     double arr[5];
//     FILE *f = fopen("setprob.data","r");

//     while ((read = getline(&line, &len, f)) != -1) 
//     {
//         p =strtok(line, " "); // get first word
//         arr[i] = strtod(p,&eptr);  // convert to double
//         i++; 
//     }
//     fclose(f);
//     free(line);

//     /* === Create and populate structures on the host === */
//     GeofloodVars geofloodVars;
//     geofloodVars.gravity = arr[0];
//     geofloodVars.dry_tolerance = arr[1];
//     geofloodVars.earth_radius = arr[2];
//     geofloodVars.coordinate_system = (int) arr[3];
//     geofloodVars.mcapa = (int) arr[4];

    
//     /* === Copy structures to device (constant memory) === */
//     CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_geofloodVars), &geofloodVars, sizeof(GeofloodVars)));
// }

// __constant__ GeofloodVars d_geofloodVars;

// void setprob_cuda() {
//     double arr[5];
//     FILE *f = fopen("setprob.data", "r");
//     if (!f) {
//         printf("setprob.data not provided");
//         return;
//     }

//     char line[256]; // Assuming each line won't exceed 256 characters
//     int i = 0;
//     while (fgets(line, sizeof(line), f)) {
//         arr[i++] = atof(line);
//         if (i >= 5) break;
//     }
//     fclose(f);

//     /* === Create and populate structures on the host === */
//     GeofloodVars geofloodVars;
//     geofloodVars.gravity = arr[0];
//     geofloodVars.dry_tolerance = arr[1];
//     geofloodVars.earth_radius = arr[2];
//     geofloodVars.coordinate_system = (int)arr[3];
//     geofloodVars.mcapa = (int)arr[4];

//     /* === Copy structures to device (constant memory) === */
//     CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_geofloodVars), &geofloodVars, sizeof(GeofloodVars)));
// }

__constant__ GeofloodVars d_geofloodVars;

void setprob_cuda() {

    /*=== declare variables === */
    int mcapa_, coord_system_;
    double grav_, dry_tol_, earth_rad_, deg2rad_;

    GET_GEOCLAW_PARAMETERS(&mcapa_,&coord_system_,&grav_,&dry_tol_, &earth_rad_,&deg2rad_);

     /* === Create and populate structures on the host === */
    GeofloodVars geofloodVars;
    geofloodVars.gravity = grav_;
    geofloodVars.dry_tolerance = dry_tol_;
    geofloodVars.earth_radius = earth_rad_;
    geofloodVars.coordinate_system = coord_system_;
    geofloodVars.mcapa = mcapa_;
    geofloodVars.deg2rad = deg2rad_;

    /* === Copy structures to device (constant memory) === */
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_geofloodVars), &geofloodVars, sizeof(GeofloodVars)));

}
