#include "hip/hip_runtime.h"
/* 
@author: David L. George
@rewritten and accelerated to CUDA by: Brian Kyanjo
@date: 31 July 2023
@description: Solves normal Riemann problems for the 2D shallow water equations (swe) with 
topography:
            h_t + (hu)_x + (hv)_y = 0
            (hu)_t + (hu^2 + 1/2gh^2)_x + (huv)_y = -ghb_x
            (hv)_t + (huv)_x + (hv^2 + 1/2gh^2)_y = -ghb_y
where h is the height, u is the x velocity, v is the y velocity, g is the gravitational constant, and b is the topography.
@input: ql - conatins the state vector at the left edge of each cell
        qr - contains the state vector at the right edge of each cell
        
        This data is along a slice in the x-direction if idir = 0 or along a slice in the y-direction if idir = 1.

        idir - indicates the direction of the slice

@note: - The ith Riemann problem has left state qr(i-1,:) and right state ql(i,:).
       - This solver allows the user to easily select a Riemann solver in riemann_solvers.c,    this routine initializes all the variables for the swe, accounting for wet dry boundary, dry cells, wave speeds, etc.
       
@reference: JCP paper by George(2008)
*/

#define maxiter 1

#include "../fc2d_cudaclaw_cuda.h"
#include "variables.h"
#include <math.h>
#include <fc2d_geoclaw.h>
#include <fc2d_cudaclaw_check.h>
// #include <fc2d_cudaclaw_options.h>
// #include <cudaclaw_user_fort.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>
#include <fclaw2d_include_all.h>

/* Extern declarations*/
extern __constant__ GeofloodVars d_geofloodVars;

/* function prototypes */
 __device__ void riemanntype(double hL, double hR, double uL, double uR, double *hm, double *s1m, double *s2m, bool *rare1, bool *rare2, double drytol);

 __device__ void riemann_aug_JCP(int meqn, int mwaves, double hL,
    double hR, double huL, double huR, double hvL, double hvR, 
    double bL, double bR, double uL, double uR, double vL, 
    double vR, double phiL, double phiR, double sE1, double sE2, double* sw, double* fw, double drytol, int idir);

/* Normal Riemann solver for the 2d shallow water equations with topography */
__device__ void cuda_flood_rpn2(int idir, int meqn, int mwaves,
                                int maux, double ql[], double qr[],
                                double auxl[], double auxr[],
                                double fwave[], double s[], 
                                double amdq[], double apdq[], double drytol, int mcapa)
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double earth_radius = d_geofloodVars.earth_radius;
    double deg2rad = d_geofloodVars.deg2rad;

    /* Local variables */
    double wall[3], fw[9], sw[3];
    double hR, hL, huR, huL, hvR, hvL, uR, uL, vR, vL, phiR, phiL;
    double bR, bL, sL, sR, sRoe1, sRoe2, sE1, sE2, uhat, chat;
    double hstar, hstartest, dxdc;
    double s1m, s2m;
    bool rare1, rare2;
    // int mw, mu, mv;

    /* === Initializing === */
    /* inform of a bad riemann problem from the start */
    // if ((qr[0] < 0.0) || (ql[0] < 0.0)) {
    //     printf("Negative input: hl, hr = %f,%f\n", ql[0], qr[0]);
    // }

    /* initialize Riemann problem for grid interface */
    for (int mw = 0; mw < mwaves; mw++){
        s[mw] = 0.0;
        fwave[mw] = 0.0;
        fwave[mw + mwaves] = 0.0;
        fwave[mw + 2*mwaves] = 0.0;
    }

    /* set normal direction */
    int mu = 1+idir;
    int mv = 2-idir;

    /* zero (small) negative values if they exist */

    if (qr[0] < 0.0) {
        qr[0] = 0.0;
        qr[1] = 0.0;
        qr[2] = 0.0;
    }

    // right state
    if (ql[0] < 0.0) {
        ql[0] = 0.0;
        ql[1] = 0.0;
        ql[2] = 0.0;
    }


    if (ql[0] > drytol || qr[0] > drytol) {
        /* Riemann problem variables */
        hL = ql[0];
        hR = qr[0];
        huL = ql[mu];
        huR = qr[mu];
        bL = auxl[0];
        bR = auxr[0];

        hvL = ql[mv];
        hvR = qr[mv];


        // Check for wet/dry left boundary
        if (hR > drytol) {
            uR = huR / hR;
            vR = hvR / hR;
            phiR = 0.5 * s_grav * (hR * hR) + (huR * huR) / hR;
        } else {
            hR = 0.0;
            huR = 0.0;
            hvR = 0.0;
            uR = 0.0;
            vR = 0.0;
            phiR = 0.0;
        }

        // Check for wet/dry right boundary
        if (hL > drytol) {
            uL = huL / hL;
            vL = hvL / hL;
            phiL = 0.5 * s_grav * (hL * hL) + (huL * huL) / hL;
        } else {
            hL  = 0.0;
            huL = 0.0;
            hvL = 0.0;
            uL  = 0.0;
            vL  = 0.0;
            phiL = 0.0;
        }

        /* left and right surfaces depth inrelation to topography */
        wall[0] = 1.0;
        wall[1] = 1.0;
        wall[2] = 1.0;
        if (hR <= drytol) {
            /* determine the wave structure */
            riemanntype(hL, hL, uL, -uL, &hstar, &s1m, &s2m, &rare1, &rare2, drytol);
            // riemann_type(hL, hL, -uL, uL, hstar, s1m, s2m, rare1, rare2, drytol);

            hstartest = fmax(hL,hstar);
            if (hstartest + bL < bR) {
                /* hL+bL < bR and hstar+bL < bR, so water can't overtop right cell 
                (move into right cell) so right state should become ghost values 
                that mirror left for wall problem) */
                wall[1] = 0.0;
                wall[2] = 0.0;
                hR = hL;
                huR = -huL;
                bR = bL;
                phiR = phiL;
                uR = -uL;
                vR = vL;
                /* here we already have huR =- huL, so we don't need to change it */
            } else if (hL+bL < bR) {
                /* hL+bL < bR and hstar+bL >bR, so we set bR to the water level in 
                the left cell so that water can possibly overtop the right cell (move into the right cell) */ 
                bR = hL + bL;
            }
        } else if (hL <= drytol) { /* right surface is lower than left topo */
            /* determine the Riemann structure */
            riemanntype(hR, hR, -uR, uR, &hstar, &s1m, &s2m, &rare1, &rare2, drytol);
            // riemann_type(hR, hR, uR, -uR, hstar, s1m, s2m, rare1, rare2, drytol);
            hstartest = fmax(hR,hstar);

            if (hstartest + bR < bL) //left state should become ghost values that mirror right for wall problem
            {
                wall[0] = 0.0;
                wall[1] = 0.0;
                hL = hR;
                huL = -huR;
                bL = bR;
                phiL = phiR;
                uL = -uR;
                vL = vR;
            } else if (hR+bR < bL) {
                bL = hR + bR;
            }
        }

        /* determine wave speeds */
        sL = uL - sqrt(s_grav*hL); // 1 wave speed of left state
        sR = uR + sqrt(s_grav*hR); // 2 wave speed of right state

        uhat = (sqrt(s_grav*hL)*uL + sqrt(s_grav*hR)*uR)/(sqrt(s_grav*hR) + sqrt(s_grav*hL)); // Roe average
        chat = sqrt(s_grav*0.5*(hR + hL)); // Roe average
        sRoe1 = uhat - chat; // Roe wave speed 1 wave
        sRoe2 = uhat + chat; // Roe wave speed 2 wave

        sE1 = fmin(sL,sRoe1); // Einfeldt wave speed 1 wave
        sE2 = fmax(sR,sRoe2); // Einfeldt wave speed 2 wave

        /* --- end of initializing --- */

        /* === solve Riemann problem === */
        riemann_aug_JCP(meqn,mwaves,hL,hR,huL,huR,hvL,hvR,bL,bR,uL,uR,vL,vR,phiL,phiR,sE1,sE2,sw,fw,drytol,idir);

        /*eliminate ghost fluxes for wall */
        int mk = 0;
        for (int mw = 0; mw < mwaves; mw++){
            /*eliminate ghost fluxes for wall*/
            sw[mw] *= wall[mw];         
            fw[mk] *= wall[mw];  mk++; 
            fw[mk] *= wall[mw];  mk++; 
            fw[mk] *= wall[mw];  mk++; 
        }

        /* update fwave and corresponding speeds */
        fwave[0]  = fw[0];
        fwave[mu] = fw[1];
        fwave[mv] = fw[2];
        s[0] = sw[0];

        fwave[mwaves + 0]  = fw[3];
        fwave[mwaves + mu] = fw[4];
        fwave[mwaves + mv] = fw[5];
        s[1] = sw[1];
       
        fwave[2*mwaves + 0]  = fw[6];
        fwave[2*mwaves + mu] = fw[7];
        fwave[2*mwaves + mv] = fw[8];
        s[2] = sw[2];

    }

    // label30: // (similar to 30 continue in Fortran)

    /* --- Capacity or Mapping from Latitude Longitude to physical space ----*/
    if (mcapa > 0) {
        // Compute dxdc based on idir without branching
        dxdc = earth_radius * deg2rad * (idir == 0 ? 1.0 : cos(auxr[2]));
    
        // Update fwave and corresponding speeds
        for (int mw = 0; mw < mwaves; mw++) {
            s[mw] *= dxdc;
            fwave[mw] *= dxdc;
            fwave[mw + mwaves] *= dxdc;
            fwave[mw + 2*mwaves] *= dxdc;
        }
    }
    
    /* --- compute fluctuations --- */
    amdq[0] = 0.0;
    amdq[1] = 0.0;
    amdq[2] = 0.0;
    apdq[0] = 0.0;
    apdq[1] = 0.0;
    apdq[2] = 0.0;
    for (int mw = 0; mw < mwaves * 3; mw++) {

        double pos_weight = s[mw / 3] > 0.0;
        double neg_weight = s[mw / 3] < 0.0;
        double zero_weight = 1.0 - pos_weight - neg_weight; // 1 if s[mw / 3]; == 0, 0 otherwise
    
        // Update amdq and apdq based on the weights
        amdq[mw % 3] += (neg_weight + 0.5 * zero_weight) * fwave[mw];
        apdq[mw % 3] += (pos_weight + 0.5 * zero_weight) * fwave[mw];
    }

}


__device__ cudaclaw_cuda_rpn2_t cudaflood_rpn2 = cuda_flood_rpn2;

void cudaflood_assign_rpn2(cudaclaw_cuda_rpn2_t *rpn2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpn2, HIP_SYMBOL(cudaflood_rpn2), sizeof(cudaclaw_cuda_rpn2_t));
    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (cuda_flood_rpn2): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}

/* Transverse Riemann solver for the 2d shallow water equations with topography 
@desc: Using The Jacobian matrix from left cell (imp == 0) or right cell (imp == 1) to compute the transverse fluxes.
*/

__device__ void cuda_flood_rpt2(int idir, int meqn, int mwaves, int maux,
                double ql[], double qr[], double aux1[], 
                double aux2[], double aux3[], int imp, 
                double asdq[], double bmasdq[], double bpasdq[], double drytol) 
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    double deg2rad = d_geofloodVars.deg2rad;

    // int mw, mu, mv;
    double s[3], beta[3], r[9];
    // double r[3][3];
    double h, u, v;
    double delf1, delf2, delf3;
    double dxdcm, dxdcp, topo1, topo3, eta;

    int mu = 1+idir;
    int mv = 2-idir;

    /* intialize  all components to 0*/
    bmasdq[0] = 0.0;
    bmasdq[mu] = 0.0;
    bmasdq[mv] = 0.0;
    bpasdq[0] = 0.0;
    bpasdq[mu] = 0.0;
    bpasdq[mv] = 0.0;

    h = (imp == 0) ? ql[0] : qr[0];

    bool debug = (idir == 0) ? 1 : 0;
  
    if (h <= drytol) return; // skip problem if dry cell (leaves bmadsq(:) = bpasdq(:) = 0)
    

    /* Compute velocities in relevant cell, and other quantities */
    int k = imp * maux; 

    // fluctuations being split is either left-going or right-going
    u = (imp == 0 ? ql[mu] : qr[mu]) / h;
    v = (imp == 0 ? ql[mv] : qr[mv]) / h;

    eta = h + aux2[k];
    topo1 = aux1[k];
    topo3 = aux3[k];

    /* Check if cell that transverse wave go into are both too high: */
    if (eta < fmin(topo1, topo3)) return; 
   
    /* Check if cell that transverse waves go into are both to high, if so,
    do the splitting (no dry cells), and compute necessary quantities */
    if (coordinate_system == 2) {
        // On the sphere
        if (idir == 1) {
            dxdcp = earth_radius * deg2rad;
            dxdcm = dxdcp;
        } else {
            
            int k = imp * maux + 2;
            dxdcp = earth_radius * cos(aux3[k]) * deg2rad;
            dxdcm = earth_radius * cos(aux1[k]) * deg2rad;
        }
    } else {
        // Cartesian
        dxdcp = 1.0;
        dxdcm = 1.0;
    }

    /* Compute some speeds necessary for the Jacobian 
    - Computing upgoing, downgoing waves either in cell on left (if imp==0)
        or on the right (if imp==1) 
    - To achieve this we use q values in cells above and below, however these
        aren't available (only in aux values)
    */
    s[0] = v - sqrt(s_grav * h);
    s[1] = v;
    s[2] = v + sqrt(s_grav * h);

    /* Determine asdq decomposition (beta) */
    delf1 = asdq[0];
    delf2 = asdq[mu];
    delf3 = asdq[mv];

    beta[0] = ((s[2]*delf1) - delf3) / (s[2] - s[0]);
    beta[1] = (-u*delf1) + delf2;
    beta[2] = (delf3 - (s[0]*delf1)) / (s[2] - s[0]);

    /* set-up eigenvectors */
    r[0] = 1.0;
    r[1] = u;
    r[2] = s[0];

    r[3] = 0.0;
    r[4] = 1.0;
    r[5] = 0.0;

    r[6] = 1.0;
    r[7] = u;
    r[8] = s[2];

    // r[0][0] = 1.0;
    // r[1][0] = u;
    // r[2][0] = s[0];

    // r[0][1] = 0.0;
    // r[1][1] = 1.0;
    // r[2][1] = 0.0;

    // r[0][2] = 1.0;
    // r[1][2] = u;
    // r[2][2] = s[2];

    /* Compute transverse fluctuations */
    int km = 0, kp = 0;
    for (int mw = 0; mw < 3; mw++) {
        // Compute condition flags
        double left_going_flag = (s[mw] < 0.0) && (eta >= topo1);
        double right_going_flag = (s[mw] > 0.0) && (eta >= topo3);
    
        // Update bmasdq and bpasdq based on conditions
        bmasdq[0]  += left_going_flag * dxdcm * s[mw] * beta[mw] * r[km]; km++;
        bmasdq[mu] += left_going_flag * dxdcm * s[mw] * beta[mw] * r[km]; km++;
        bmasdq[mv] += left_going_flag * dxdcm * s[mw] * beta[mw] * r[km]; km++;
    
        bpasdq[0]  += right_going_flag * dxdcp * s[mw] * beta[mw] * r[kp]; kp++;
        bpasdq[mu] += right_going_flag * dxdcp * s[mw] * beta[mw] * r[kp]; kp++;
        bpasdq[mv] += right_going_flag * dxdcp * s[mw] * beta[mw] * r[kp]; kp++;

        // bmasdq[0]  += left_going_flag * dxdcm * s[mw] * beta[mw] * r[0][mw];
        // bmasdq[mu] += left_going_flag * dxdcm * s[mw] * beta[mw] * r[1][mw];
        // bmasdq[mv] += left_going_flag * dxdcm * s[mw] * beta[mw] * r[2][mw];
    
        // bpasdq[0]  += right_going_flag * dxdcp * s[mw] * beta[mw] * r[0][mw];
        // bpasdq[mu] += right_going_flag * dxdcp * s[mw] * beta[mw] * r[1][mw];
        // bpasdq[mv] += right_going_flag * dxdcp * s[mw] * beta[mw] * r[2][mw];
    }
    
}



__device__ cudaclaw_cuda_rpt2_t cudaflood_rpt2 = cuda_flood_rpt2;

void cudaflood_assign_rpt2(cudaclaw_cuda_rpt2_t *rpt2)
{
    hipError_t ce = hipMemcpyFromSymbol(rpt2, HIP_SYMBOL(cudaflood_rpt2), sizeof(cudaclaw_cuda_rpt2_t));

    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (cuda_flood_rpt2): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}

/* === Begin fuction riemann_aug_JCP======================================================== @description: - Solves swe give single left and right states
@note: - To use the original solver call with maxiter=1.
       - This solver allows iteration when maxiter > 1. The iteration seems to help  
         with instabilities that arise (with any solver) as flow becomes transcritical 
         over variable topography due to loss of hyperbolicity. 
*/

__device__ void riemann_aug_JCP(int meqn, int mwaves, double hL,
    double hR, double huL, double huR, double hvL, double hvR, 
    double bL, double bR, double uL, double uR, double vL, 
    double vR, double phiL, double phiR, double sE1, double sE2, double* sw, double* fw, double drytol, int idir)
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;

    /* Local variables */
    // double A[9], r[9], lambda[3], del[3], beta[3];
    double lambda[3], beta[3],del[3];
    // double A[3][3], r[3][3];
    double A[9], r[9];
    double delh, delhu, delphi, delb, delnorm;
    double rare1st, rare2st, sdelta, raremin, raremax;
    double criticaltol, convergencetol;
    double criticaltol_2, hustar_interface;
    double s1s2bar, s1s2tilde, hbar, hLstar, hRstar;
    double huRstar, huLstar, uRstar, uLstar, hstarHLL;
    double deldelh, deldelphi;
    double s1m, s2m, hm;
    double det1, det2, det3, determinant;
    bool rare1, rare2, rarecorrector, rarecorrectortest, sonic;
    // int mw, k, iter;

    /* determine del vectors */
    delh = hR - hL;
    delhu = huR - huL;
    delphi = phiR - phiL;
    delb = bR - bL;
    delnorm = delh * delh + delphi * delphi;

    /* Determine the Riemann structure */
    riemanntype(hL,hR,uL,uR,&hm,&s1m,&s2m,&rare1,&rare2,drytol);
    // riemann_type(hL,hR,uL,uR,hm,s1m,s2m,rare1,rare2, drytol);

    /* For the solver to handle depth negativity, depth dh is included in the decompostion which gives as acess to using the depth positive semidefinite solver (HLLE). This makes the system to have 3 waves instead of 2. where the 1st and 3rd are the eigenpairs are related to the flux Jacobian matrix of the original SWE (since s1<s2<s3, and have been modified by Einfeldt to handle depth non-negativity) and the 2nd is refered to as the the entropy corrector wave since its introduced to correct entropy violating solutions with only 2 waves. */
    
    /* The 1st and 3rd speeds are the eigenvalues of the Jacobian matrix of the original SWE modified by Einfeldt's for use with the HLLE solver. */
    lambda[0] = fmin(sE1, s2m); /* sE1 - flux Jacobian eigen value s2m - Roe speed */
    lambda[2] = fmax(sE2, s1m); /* sE2 - flux Jacobian eigen value s1m - Roe speed*/

    /* Einfeldt's speeds */
    sE1 = lambda[0]; 
    sE2 = lambda[2];

    /* The 2nd speed is the entropy corrector wave speed. */
    lambda[1] = 0.0; /* no strong or significant rarefaction waves */
    
    /* determine the middle state in the HLLE solver */
    hstarHLL = fmax((hL*uL - hR*uR + (sE2 * hR) - (sE1 * hL)) / (sE2 - sE1), 0.0); /* middle state between the two discontinuities (positive semidefinite depth) */

    /* === determine the middle entropy corrector wave === */
    /* rarecorrectortest = .true. provides a more accurate Riemann solution but is more expensive. This is because a nonlinear Riemann solution with  2 nonlinear waves as a linear Riemann solution 3 (or 2 jump discontionuities to approximate 1 smooth nonlinear rarefaction if it's large). When rarecorrectortest = .false. the approximate solution has only 2 jump discontinuities instead of 3, so its less accurate but faster. */
    rarecorrectortest = false;
    rarecorrector = false;
    if (rarecorrectortest) {
        sdelta = lambda[2] - lambda[0];
        raremin = 0.5; /* indicate a large rarefaction wave but not large */
        raremax = 0.9; /* indicate a very large rarefaction wave */
       /* i.e (the total speed difference between the fastest and slowest wave in the Riemann solution = 0.5) */

        if (rare1 && sE1 * s1m < 0.0) raremin = 0.2;
        if (rare2 && sE2 * s2m < 0.0) raremin = 0.2;

        if (rare1 || rare2) {
            /* check which rarefaction is the strongest */
            rare1st = 3.0 * (sqrt(s_grav * hL) - sqrt(s_grav * hm));
            rare2st = 3.0 * (sqrt(s_grav * hR) - sqrt(s_grav * hm));
            if (fmax(rare1st, rare2st) > raremin * sdelta && fmax(rare1st, rare2st) < raremax * sdelta) {
                rarecorrector = true;
                if (rare1st > rare2st) {
                    lambda[1] = s1m;
                } else if (rare2st > rare1st) {
                    lambda[1] = s2m;
                } else {
                    lambda[1] = 0.5 * (s1m + s2m);
                }
            }
        }
        if (hstarHLL < fmin(hL, hR) / 5.0) rarecorrector = false;
    }

    /* determining modified eigen vectors */
    int k = 0;
    for (int mw = 0; mw < mwaves; mw++) {   
        r[k] = 1.0; k++;
        r[k] = lambda[mw]; k++;
        r[k] = lambda[mw]*lambda[mw]; k++;
       
        // r[0][mw] = 1.0;
        // r[1][mw] = lambda[mw];
        // r[2][mw] = pow(lambda[mw],2.0);
    }

    /* no strong rarefaction wave */
    if (!rarecorrector) {
        lambda[1]= 0.5*(lambda[0] + lambda[2]);
        r[3] = 0.0;
        r[4] = 0.0;
        r[5] = 1.0;
        // r[0][1] = 0.0;
        // r[1][1] = 0.0;
        // r[2][1] = 1.0;
    }

    /* === Determine the steady state wave === */
    criticaltol = fmax(drytol*s_grav, 1.0e-6);
    criticaltol_2 = sqrt(criticaltol);
    deldelh = -delb;
    deldelphi = -0.5 * (hR + hL) * (s_grav * delb); /* some approximation of the source term \int_{x_{l}}^{x_{r}} -g h b_x dx */

    /* determine a few quantities needed for steady state wave if iterated */
    hLstar = hL;
    hRstar = hR;
    uLstar = uL;
    uRstar = uR;
    huLstar = uLstar * hLstar;
    huRstar = uRstar * hRstar;

    /* iterate to better find the steady state wave */
    convergencetol = 1e-6;
    for (int iter=1; iter <= maxiter; iter++) {
        /* determine steady state wave (this will be subtracted from the delta vectors */
        if (fmin(hLstar,hRstar) < drytol && rarecorrector) {
            rarecorrector = false;
            hLstar = hL;
            hRstar = hR;
            uLstar = uL;
            uRstar = uR;
            huLstar = uLstar*hLstar;
            huRstar = uRstar*hRstar;
            lambda[1] = 0.5*(lambda[0] + lambda[2]);
            r[3] = 0.0;
            r[4] = 0.0;
            r[5] = 1.0;

            // r[0][1] = 0.0;
            // r[1][1] = 0.0;
            // r[2][1] = 1.0;
        }

        /* For any two states; Q_i and Q_i-1, eigen values of SWE must satify: lambda(q_i)*lambda(q_i-1) = u^2 -gh, writing this conditon as a function of Q_i and Q_i-1, u and h become averages in lambda(q_i)*lambda(q_i-1) = u^2 -gh and these averages are denoted by bar and tilde. */
        hbar = fmax(0.5 * (hLstar + hRstar), 0.0);
        s1s2bar = 0.25 * (uLstar + uRstar)*(uLstar + uRstar) - (s_grav * hbar);
        s1s2tilde = fmax(0.0, uLstar * uRstar) - (s_grav * hbar);

        /* Based on the above conditon, smooth staedy state over slopping bathymetry cannot have a sonic point. Therefore, for regions with monotonically varying bathymetry, steady-state flow is either entirely subsonic (-u^2 +gh > 0) or entirely supersonic. */

        sonic = (fabs(s1s2bar) <= criticaltol) ||
                (s1s2bar * s1s2tilde <= criticaltol * criticaltol) ||
                (s1s2bar * sE1 * sE2 <= criticaltol * criticaltol) ||
                (fmin(fabs(sE1), fabs(sE2)) < criticaltol_2) ||
                (sE1 < criticaltol_2 && s1m > -criticaltol_2) ||
                (sE2 > -criticaltol_2 && s2m < criticaltol_2) ||
                ((uL + sqrt(s_grav * hL)) * (uR + sqrt(s_grav * hR)) < 0.0) ||
                ((uL - sqrt(s_grav * hL)) * (uR - sqrt(s_grav * hR)) < 0.0);

        /* find jump in h, deldelh */
        deldelh = sonic ? -delb : delb * s_grav * hbar / s1s2bar;
    
        /* find bounds in case of critical state resonance, or negative states */
        // Calculate condition flags as 0 or 1
        double condition1 = (sE1 < -criticaltol) && (sE2 > criticaltol);
        double condition2 = (sE1 >= criticaltol);
        double condition3 = (sE2 <= -criticaltol);

        // Compute potential updates to deldelh under each condition
        double update1_min = hstarHLL * (sE2 - sE1) / sE2;
        double update1_max = hstarHLL * (sE2 - sE1) / sE1;
        double update2_min = hstarHLL * (sE2 - sE1) / sE1;
        double update2_max = -hL;
        double update3_min = hR;
        double update3_max = hstarHLL * (sE2 - sE1) / sE2;

        // Apply updates based on conditions
        deldelh = fmin(deldelh, condition1 * update1_min + condition2 * update2_min + condition3 * update3_min + (!condition1 && !condition2 && !condition3) * deldelh);
        deldelh = fmax(deldelh, condition1 * update1_max + condition2 * update2_max + condition3 * update3_max + (!condition1 && !condition2 && !condition3) * deldelh);

        /* find jump in phi, ddphi */
        deldelphi = -delb * s_grav * hbar * (sonic ? 1.0 : s1s2tilde / s1s2bar);

        /* find bounds in case of critical state resonance, or negative states */
        deldelphi = fmin(deldelphi, s_grav * fmax(-hLstar * delb, -hRstar * delb));
        deldelphi = fmax(deldelphi, s_grav * fmin(-hLstar * delb, -hRstar * delb));

        /* determine the delta vectors */
        del[0] = delh - deldelh;
        del[1] = delhu;
        del[2] = delphi - deldelphi;  

        /* Determine coefficients beta(k) using crammer's rule
          first determine the determinant of the eigenvector matrix */
        // det1 = r[0][0]*(r[1][1]*r[2][2] - r[1][2]*r[2][1]);
        // det2 = r[0][1]*(r[1][0]*r[2][2] - r[1][2]*r[2][0]);
        // det3 = r[0][2]*(r[1][0]*r[2][1] - r[1][1]*r[2][0]);
        det1 = r[0]*(r[4]*r[8] - r[7]*r[5]);
        det2 = r[3]*(r[1]*r[8] - r[7]*r[2]);
        det3 = r[6]*(r[1]*r[5] - r[4]*r[2]);
        determinant = det1 - det2 + det3;

        /* solve for beta(k) */
        int kk = 0;
        for (int k = 0; k < 3; k++) {
            // Copy the entire matrix r into A for each iteration
            // A[0][0] = r[0][0];
            // A[0][1] = r[0][1];
            // A[0][2] = r[0][2];
            // A[1][0] = r[1][0];
            // A[1][1] = r[1][1];
            // A[1][2] = r[1][2];
            // A[2][0] = r[2][0];
            // A[2][1] = r[2][1];
            // A[2][2] = r[2][2];

            A[0] = r[0]; A[1] = r[1]; A[2] = r[2];
            A[3] = r[3]; A[4] = r[4]; A[5] = r[5];
            A[6] = r[6]; A[7] = r[7]; A[8] = r[8];
        
            // Modify the k-th column of A
            // A[0][k] = del[0];
            // A[1][k] = del[1];
            // A[2][k] = del[2];
            A[kk] = del[0]; kk++;
            A[kk] = del[1]; kk++;
            A[kk] = del[2]; kk++;
        
            // Calculate the determinant components
            // double det1 = A[0][0] * (A[1][1] * A[2][2] - A[1][2] * A[2][1]);
            // double det2 = A[0][1] * (A[1][0] * A[2][2] - A[1][2] * A[2][0]);
            // double det3 = A[0][2] * (A[1][0] * A[2][1] - A[1][1] * A[2][0]);
            det1 = A[0]* (A[4]*A[8] - A[7]*A[5]);
            det2 = A[3]* (A[1]*A[8] - A[7]*A[2]);
            det3 = A[6]* (A[1]*A[5] - A[4]*A[2]);
        
            // Compute the final value for this iteration
            beta[k] = (det1 - det2 + det3) / determinant;
        }
        

        /* exit if things aren't changing */
        // if (fabs(pow(del[0],2)+pow(del[2],2.0) - delnorm) < convergencetol) break;
        if (fabs(del[0]*del[0] + del[2]*del[2] - delnorm) < convergencetol) break;
        // if (fabs(pow(del[0],2)+pow(del[2],2.0) - delnorm) < convergencetol) return;


        delnorm = del[0]*del[0] + del[2]*del[2]; /* update delnorm */

        /* find new states qLstar and qRstar on either side of interface */
        hLstar = hL;
        hRstar = hR;
        uLstar = uL;
        uRstar = uR;
        huLstar = uLstar*hLstar;
        huRstar = uRstar*hRstar;

        /* left state depth and momentum updates */
        // k = 0;
        // for (int mw = 0; mw < mwaves; mw++) {
        //     double multiplier = lambda[mw] < 0.0 ? 1.0 : 0.0;
        //     // hLstar += multiplier * beta[mw] * r[0][mw];
        //     // huLstar += multiplier * beta[mw] * r[1][mw];

        //     hLstar  += multiplier * beta[mw] * r[k]; k++;
        //     huLstar += multiplier * beta[mw] * r[k]; k=k+2;
        // }
        
        int k=0;
        for (int mw=0; mw < mwaves; mw++)
        {
            if (lambda[mw] < 0.0)
            {
               hLstar = hLstar + beta[mw]*r[k]; k++;
               huLstar = huLstar + beta[mw]*r[k]; k=k+2;
            }
        }

        /* right state depth and momentum updates */
        // k = 0;
        // for (int mw = mwaves - 1; mw >= 0; mw--) {
        //     double multiplier = lambda[mw] > 0.0 ? 1.0 : 0.0;
        //     // hRstar -= multiplier * beta[mw] * r[0][mw];
        //     // huRstar -= multiplier * beta[mw] * r[1][mw];

        //     hRstar  -= multiplier * beta[mw] * r[k]; k++;
        //     huRstar -= multiplier * beta[mw] * r[k]; k=k+2;
        // }
    
        int kw = 0;
        for (int mw = mwaves-1; mw >= 0; mw--)
        {
            if (lambda[mw] > 0.0)
            { 
                hRstar = hRstar - beta[mw]*r[kw]; kw++;;
                huRstar = huRstar - beta[mw]*r[kw]; kw=kw+2;
            }
        }

        /* left state velocity update */
        // hLstar = fmax(hLstar, 0.0); // Ensure hLstar is non-negative
        // uLstar = hLstar > drytol ? huLstar / hLstar : 0.0; // Update uLstar: set to huLstar/hLstar if hLstar > drytol, else to 0.0

        if (hLstar > drytol) 
        {
            uLstar = huLstar/hLstar;
        }
        else  /* dry state */
        {
            hLstar = fmax(hLstar,0.0);
            uLstar = 0.0;
        }

        /* right state velocity update */
        // hRstar = fmax(hRstar, 0.0); // Ensure hRstar is non-negative
        // uRstar = hRstar > drytol ? huRstar / hRstar : 0.0; // // Update uRstar: set to huRstar/hRstar if hRstar > drytol, else to 0.0

        if (hRstar > drytol) 
        {
            uRstar = huRstar/hRstar;
        }
        else /* dry state */
        {
            hRstar = fmax(hRstar,0.0);
            uRstar = 0.0;
        }
    } /* end of  iteration on the Riemann problem*/

    /* === determine the fwaves and speeds=== */
    int kf = 0; int kr = 1;
    for(int mw=0; mw<mwaves; mw++) {
        sw[mw] = lambda[mw];
        // fw[k]  = beta[mw] * r[1][mw]; k++;
        // fw[k]  = beta[mw] * r[2][mw]; k++;
        // fw[k]  = beta[mw] * r[1][mw]; k++;

        fw[kf] = beta[mw] * r[kr]; kf++;  
        fw[kf] = beta[mw] * r[kr+1]; kf++; 
        fw[kf] = beta[mw] * r[kr]; kf++; 
        kr += mwaves; 
    }

    // find transverse components (ie huv jumps)
    fw[2] *= vL;
    fw[8] *= vR;
    fw[5] = 0.0;

    hustar_interface = hL*uL + fw[0];
    int indexToUpdate = hustar_interface <= 0.0 ? 2 : 8;
    fw[indexToUpdate] += (hR * uR * vR - hL * uL * vL - fw[2] - fw[8]);

    // hustar_interface = hL*uL + fw[0];
    // if (hustar_interface <= 0.0) {
    //     fw[mv] += (hR * uR * vR - hL * uL * vL - fw[mv] - fw[2*mwaves + mv]);
    // } else {
    //     fw[2*mwaves + mv] += (hR * uR * vR - hL * uL * vL - fw[mv] - fw[2*mwaves + mv]);
    // }
}

/* === Begin fuction Riemann type ============
 @description: Determines the Riemann structure (wave-type in each family)
*/

__device__ void riemanntype(double hL, double hR, double uL, double uR, double *hm, 
                            double *s1m, double *s2m, bool *rare1, bool *rare2, double drytol)
{
    /* Access the __constant__ variables in variables.h */
    double s_grav = d_geofloodVars.gravity;

    // Local variables
    double u1m, u2m, h0, F_max, F_min, dfdh, F0, slope, gL, gR;
    double sqrtgh1, sqrtgh2;
    // double um;
    int iter; 

    // Test for Riemann structure
    double h_min = fmin(hR,hL);
    double h_max = fmax(hR,hL);
    double delu = uR - uL;

    /* Have dry state on either side 
    - Only one rarefaction wave
    - another shock wave has 0 jump and moves at the same speed as one edge of the    rarefaction  wave */
    if (h_min <= drytol)
    {
        *hm = 0.0;
        // um = 0.0;
    
        /* Either hR or hL is almost zero, so the expression below corresponds
           to either Eqn. (54a) or Eqn. (54b) in the JCP paper */
        *s1m = uR + uL - 2.0 * sqrt(s_grav * hR) + 2.0 * sqrt(s_grav * hL);
        *s2m = uR + uL - 2.0 * sqrt(s_grav * hR) + 2.0 * sqrt(s_grav * hL); 
        *rare1 = (hL <= 0.0) ? false : true;
        *rare2 = !(*rare1);
    } else {
        F_min = delu + 2.0 * (sqrt(s_grav * h_min) - sqrt(s_grav * h_max));
        F_max = delu + (h_max - h_min) * sqrt(0.5 * s_grav * (h_max + h_min) / (h_max * h_min));

        if (F_min > 0.0){  // 2-rarefactions
            /* Eqn (13.56) in the FVMHP book */
            double hm_flag = fmax(0.0, -delu + 2.0 * (sqrt(s_grav * hL) + sqrt(s_grav * hR)));
            *hm = (1.0 / (16.0 * s_grav)) * hm_flag * hm_flag;
            // um = copysign(1.0, *hm) * (uL + 2.0 * (sqrt(s_grav * hL) - sqrt(s_grav * *hm)));
            *s1m = uL + 2.0 * sqrt(s_grav * hL) - 3.0 * sqrt(s_grav * *hm);
            *s2m = uR - 2.0 * sqrt(s_grav * hR) + 3.0 * sqrt(s_grav * *hm);
            *rare1 = true;
            *rare2 = true;
        } else if (F_max <= 0.0) { // 2-shocks
            /* Below it solves for the intersection of two Hugoniot loci to get the
            accurate Riemann solution */
            /* Root finding using a Newton iteration on sqrt(h) */
            h0 = h_max;
            for (iter = 1; iter <= maxiter; iter++) {
                gL = sqrt(0.5 * s_grav * (1.0 / h0 + 1.0 / hL));
                gR = sqrt(0.5 * s_grav * (1.0 / h0 + 1.0 / hR));
                F0 = delu + (h0 - hL) * gL + (h0 - hR) * gR;
                dfdh = gL - s_grav * (h0 - hL) / (4.0 * h0 * h0 * gL) + gR - s_grav * (h0 - hR) / (4.0 * h0 * h0 * gR);
                slope = 2.0 * sqrt(h0) * dfdh;
                h0 = (sqrt(h0) - (F0 / slope))*(sqrt(h0) - (F0 / slope));
            }
            *hm = h0;
            /* u1m and u2m are Eqns (13.19) and (13.20) in the FVMHP book */
            u1m = uL - (*hm - hL) * sqrt(0.5 * s_grav * (1.0 / *hm + 1.0 / hL));
            u2m = uR + (*hm - hR) * sqrt(0.5 * s_grav * (1.0 / *hm + 1.0 / hR));
            // um = 0.5 * (u1m + u2m);
            *s1m = u1m - sqrt(s_grav * *hm);
            *s2m = u2m + sqrt(s_grav * *hm);
            *rare1 = false;
            *rare2 = false;
        } else { // 1-shock or 1-rarefaction
            h0 = h_min;
            for (iter = 1; iter <= maxiter; iter++) {
                F0 = delu + 2.0 * (sqrt(s_grav * h0) - sqrt(s_grav * h_max)) + (h0 - h_min) * sqrt(0.5 * s_grav * (1.0 / h0 + 1.0 / h_min));
                slope = (F_max - F0) / (h_max - h_min);
                h0 = h0 - F0 / slope;
            }

            *hm = h0;
            // sqrtgh2 = sqrt(s_grav * *hm);
            // sqrtgh1 = sqrt(s_grav * (hL > hR ? hL : hR));

            //  /* These two equations are extracted from Eqn (13.55) in the FVMHP book */
            // *s1m =  hL > hR ? (uL + 2.0 * sqrtgh1 - 3.0 * sqrtgh2) : (uR - 2.0 * sqrtgh1 + sqrtgh2);
            // *s2m =  hL > hR ? (uL + 2.0 * sqrtgh1 - sqrtgh2) : (uR - 2.0 * sqrtgh1 + 3.0 * sqrtgh2);

            // *rare1 =  hL > hR;
            // *rare2 = !(*rare1);

            sqrtgh2 = sqrt(s_grav * *hm);
            if (hL > hR) {
                sqrtgh1 = sqrt(s_grav * hL);
                /* Eqn (13.55) in the FVMHP book */
                // um = uL + 2.0 * sqrtgh1 - 2.0 * sqrtgh2;
                *s1m = uL + 2.0 * sqrtgh1 - 3.0 * sqrtgh2;
                *s2m = uL + 2.0 * sqrtgh1 - sqrtgh2;

                *rare1 = true;
                *rare2 = false;
            } else {
                sqrtgh1 = sqrt(s_grav * hR);
                // um = uR - 2.0 * sqrtgh1 + 2.0 * sqrtgh2;
                *s1m = uR - 2.0 * sqrtgh1 + sqrtgh2;
                *s2m = uR - 2.0 * sqrtgh1 + 3.0 * sqrtgh2;
                *rare1 = false;
                *rare2 = true;
            }
        }
    }
} /* End of riemanntype function */

