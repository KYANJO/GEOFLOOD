#include "hip/hip_runtime.h"
/* 
    @author Brian Kyanjo briankyanjo@u.boisestate.edu
    @date 2024.03.07
    @brief src2 kernel function for GeoFlood

*/

#include "../fc2d_cudaclaw_cuda.h"
#include "variables.h"
#include <math.h>
#include <fc2d_geoclaw.h>
#include <fc2d_cudaclaw_check.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>
#include <fclaw2d_include_all.h>

/* Extern declarations*/
extern __constant__ GeofloodVars d_geofloodVars;


/* ================= function prototypes ================= */
__device__ double coriolis(double y);


/* ================= src2 Kernel function ================= */
// @desc: src2 kernel function for GeoFlood, contains friction and coriolis source terms
__device__ void cuda_flood_src2(int meqn, int maux, double xlower, double ylower, double dx, 
                    double dy,    double *qr, double *auxr, double t, double dt, int i, int j)
{
    double g = d_geofloodVars.gravity;
    double dry_tolerance = d_geofloodVars.dry_tolerance;
    double earth_radius = d_geofloodVars.earth_radius;
    int coordinate_system = d_geofloodVars.coordinate_system;
    double deg2rad = d_geofloodVars.deg2rad;
    bool coriolis_forcing = d_geofloodVars.coriolis_forcing;
    bool friction_forcing = d_geofloodVars.friction_forcing;
    double friction_depth = d_geofloodVars.friction_depth;
    bool variable_friction = d_geofloodVars.variable_friction;
    int num_manning = d_geofloodVars.num_manning;
    int friction_index = d_geofloodVars.friction_index;
    double manning_coefficient = d_geofloodVars.manning_coefficent;
    double manning_break = d_geofloodVars.manning_break;


    // local variables
    double depth_tolerance = 1.0e-30;
    
    // ------------- Friction source term -----------------
    if (friction_forcing)
    {
        // Extract approximate momentum
        if (qr[0] < depth_tolerance)
        {
            qr[1] = 0.0;
            qr[2] = 0.0;
        }
        else
        {
            // Apply friction source term only if in shallower water
            double coeff;
            if (qr[0] <= friction_depth)
            {
                if (!(variable_friction))
                {
                    // for (int nman = num_manning - 1; nman >= 0; nman--) 
                    {
                        // if (auxr[0] < manning_break[nman])
                        if (auxr[0] < manning_break)
                        {
                            // coeff = manning_coefficient[nman];
                            coeff = manning_coefficient;
                        }
                    }
                }
                else
                {
                    coeff = auxr[friction_index];
                }

                // Apply friction source term
                double gamma =  sqrt(pow(qr[1],2) + pow(qr[2],2))* g * pow(coeff,2) / (pow(qr[0],7/3));
                double dgamma = 1.0 + dt * gamma;
                // printf("dgamma: %f\n", dgamma);
                qr[1] = qr[1] / dgamma;
                qr[2] = qr[2] / dgamma;
            }
        }
    }
    // ------------- End of friction source term -----------------

    // ------------- Coriolis source term -----------------------
    // printf("Coriolis source term\n", coriolis_forcing);
    if (coriolis_forcing)
    {
        double y = ylower + (j - 0.5) * dy;
        double fdt = coriolis(y) * dt; // Calculate f dependent on coordinate system

        // calculate the matrix components
        double a[2][2];
        a[0][0] = 1.0 - (0.5*fdt*fdt) + pow(fdt,4)/24.0;
        a[0][1] = fdt - pow(fdt,3)/6.0;
        a[1][0] = -fdt + pow(fdt,3)/6.0;
        a[1][1] = a[0][0]; 

        // ??
        qr[1] = qr[1] * a[0][0] + qr[2] * a[0][1];
        qr[2] = qr[1] * a[1][0] + qr[2] * a[1][1];
    }
    // ------------- End of Coriolis source term -----------------
}

__device__ cudaclaw_cuda_src2_t cudaflood_src2 = cuda_flood_src2;

void cudaflood_assign_src2(cudaclaw_cuda_src2_t *src2)
{
    hipError_t ce = hipMemcpyFromSymbol(src2, HIP_SYMBOL(cudaflood_src2), sizeof(cudaclaw_cuda_src2_t));
    if(ce != hipSuccess)
    {
        fclaw_global_essentialf("ERROR (cuda_flood_src2): %s\n",hipGetErrorString(ce));
        exit(0);
    }
}

/* Calculate the coriolis constant f 
- if coordinate system  == 1 (Cartesian) then 
    A beta-plane approximation is used and y should be in meters
- if coordinate system == 2 (spherical) then
    Grid is in lat-lon coordinates and y should be in degrees which
    is then converted to radians
*/
__device__ double coriolis(double y)
{
    int coordinate_system = d_geofloodVars.coordinate_system;
    double deg2rad = d_geofloodVars.deg2rad;
    double theta_0 = d_geofloodVars.theta_0;
    double omega = d_geofloodVars.omega;

    // Assume beta-plane approximation and y in meters
    if (coordinate_system == 1)
    {
        double theta = y / 111000.0 * deg2rad + theta_0;
        return 2.0 * omega * (sin(theta_0) + (theta - theta_0) * cos(theta_0));
    }
    else if (coordinate_system == 2)
    {
        return 2.0 * omega * sin(y*deg2rad);
    }
    else
    {
        // Unknown coordinate system, return 0.0
        return 0.0;
    }
}